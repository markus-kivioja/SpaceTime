#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"

constexpr double CREATION_RAMP_START = 0.1;
constexpr double EXPANSION_START = CREATION_RAMP_START + 10.5; // When the expansion starts in ms

//#include "AliceRingRamps.h"
#include "KnotRamps.h"

std::string getProjectionString()
{
#if BASIS == X_QUANTIZED
	return "proj_x";
#elif BASIS == Y_QUANTIZED
	return "proj_y";
#elif BASIS == Z_QUANTIZED
	return "proj_z";
#endif
}

#include "Output/Picture.hpp"
#include "Output/Text.hpp"
#include "Types/Complex.hpp"
#include "Mesh/DelaunayMesh.hpp"

#include <iostream>
#include <sstream>
#include <chrono>
#include <random>

#include "mesh.h"

#define RELATIVISTIC 1

#define COMPUTE_GROUND_STATE 0

#define SAVE_STATES 0
#define SAVE_PICTURE 1

#define THREAD_BLOCK_X 16
#define THREAD_BLOCK_Y 2
#define THREAD_BLOCK_Z 1

constexpr double DOMAIN_SIZE_X = 24.0;
constexpr double DOMAIN_SIZE_Y = 24.0;
constexpr double DOMAIN_SIZE_Z = 24.0;

constexpr double REPLICABLE_STRUCTURE_COUNT_X = 112.0;
//constexpr double REPLICABLE_STRUCTURE_COUNT_Y = 112.0;
//constexpr double REPLICABLE_STRUCTURE_COUNT_Z = 112.0;

constexpr double N = 2e5; // Number of atoms in the condensate

constexpr double trapFreq_r = 126;
constexpr double trapFreq_z = 166;

constexpr double omega_r = trapFreq_r * 2 * PI;
constexpr double omega_z = trapFreq_z * 2 * PI;
constexpr double lambda_x = 1.0;
constexpr double lambda_y = 1.0;
constexpr double lambda_z = omega_z / omega_r;

constexpr double a_bohr = 5.2917721092e-11; //[m] Bohr radius
constexpr double a_0 = 101.8;
constexpr double a_2 = 100.4;

constexpr double atomMass = 1.44316060e-25;
constexpr double hbar = 1.05457148e-34; // [m^2 kg / s]
const double a_r = sqrt(hbar / (atomMass * omega_r)); //[m]

const double c0 = 4 * PI * N * (a_0 + 2 * a_2) * a_bohr / (3 * a_r);
const double c2 = 4 * PI * N * (a_2 - a_0) * a_bohr / (3 * a_r);

constexpr double myGamma = 2.9e-30;
const double alpha = N * N * myGamma * 1e-12 / (a_r * a_r * a_r * a_r * a_r * a_r * 2 * PI * trapFreq_r);

constexpr double muB = 9.27400968e-24; // [m^2 kg / s^2 T^-1] Bohr magneton

const double BqScale = -(0.5 * muB / (hbar * omega_r) * a_r) / 100.; // [cm/Gauss]
constexpr double BzScale = -(0.5 * muB / (hbar * omega_r)) / 10000.; // [1/Gauss]

constexpr double A_hfs = 3.41734130545215;
const double BqQuadScale = 100 * a_r * sqrt(0.25 * 1000 * (1.399624624 * 1.399624624) / (trapFreq_r * 2 * A_hfs)); //[cm/Gauss]
const double BzQuadScale = sqrt(0.25 * 1000 * (1.399624624 * 1.399624624) / (trapFreq_r * 2 * A_hfs)); //[1/Gauss]  \sqrt{g_q}

constexpr double SQRT_2 = 1.41421356237309;
constexpr double INV_SQRT_2 = 0.70710678118655;

constexpr double NOISE_AMPLITUDE = 0.1;

#if RELATIVISTIC
double dt = 3e-3; // Max hyperbolic: 3e-3
#else
double dt = 7e-4; // Max parabolic: 7e-4
#endif

const float IMAGE_SAVE_INTERVAL = 0.5; // ms
uint IMAGE_SAVE_FREQUENCY = uint(IMAGE_SAVE_INTERVAL * 0.5 / 1e3 * omega_r / dt) + 1;

const uint STATE_SAVE_INTERVAL = 10.0; // ms

double t = 0; // Start time in ms
double END_TIME = 0.6; // End time in ms

#if RELATIVISTIC
double sigma = 1.0; // 0.01;
double dt_per_sigma = dt / sigma;
#else
double sigma = 0.0;
double dt_per_sigma = 1.0;
#endif

enum class Phase
{
	Polar = 0,
	Ferromagnetic
};
constexpr Phase initPhase = Phase::Polar;
//constexpr Phase initPhase = Phase::Ferromagnetic;

std::string toStringShort(const double value)
{
	std::ostringstream out;
	out.precision(2);
	out << std::fixed << value;
	return out.str();
};

#if RELATIVISTIC
const std::string GROUND_STATE_PSI_FILENAME = "ground_state_psi_" + toStringShort(sigma) + ".dat";
const std::string GROUND_STATE_Q_FILENAME = "ground_state_q_" + toStringShort(sigma) + ".dat";
#else
const std::string GROUND_STATE_PSI_FILENAME = "ground_state.dat";
#endif

__device__ __inline__ double trap(double3 p)
{
	double x = p.x * lambda_x;
	double y = p.y * lambda_y;
	double z = p.z * lambda_z;
	return 0.5 * (x * x + y * y + z * z) + 100.0;
}

__device__ __inline__ double3 magneticField(double3 p, double Bq, double3 Bb)
{
	return { Bq * p.x + Bb.x, Bq * p.y + Bb.y, -2 * Bq * p.z + Bb.z };
}

__global__ void maxHamilton(double* maxHamlPtr, PitchedPtr prevStep, MagFields Bs, uint3 dimensions, double block_scale, double3 p0, double c0, double c2)
{
	const size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	const size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	const size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	// Calculate the pointers for this block
	char* prevPsi = prevStep.ptr + prevStep.slicePitch * zid + prevStep.pitch * yid + sizeof(BlockPsis) * dataXid;

	// Update psi
	const size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)
	const Complex3Vec prev = ((BlockPsis*)prevPsi)->values[dualNodeId];

	const double normSq_s1 = prev.s1.x * prev.s1.x + prev.s1.y * prev.s1.y;
	const double normSq_s0 = prev.s0.x * prev.s0.x + prev.s0.y * prev.s0.y;
	const double normSq_s_1 = prev.s_1.x * prev.s_1.x + prev.s_1.y * prev.s_1.y;
	const double normSq = normSq_s1 + normSq_s0 + normSq_s_1;

	const double3 localPos = d_localPos[dualNodeId];
	const double3 globalPos = { p0.x + block_scale * (dataXid * BLOCK_WIDTH_X + localPos.x),
		p0.y + block_scale * (yid * BLOCK_WIDTH_Y + localPos.y),
		p0.z + block_scale * (zid * BLOCK_WIDTH_Z + localPos.z) };
	const double totalPot = trap(globalPos) + c0 * normSq;

	double3 hamilton = { totalPot, totalPot, totalPot };

	const double2 temp = SQRT_2 * (conj(prev.s1) * prev.s0 + conj(prev.s0) * prev.s_1);
	const double3 magnetization = { temp.x, temp.y, normSq_s1 - normSq_s_1 };
	double3 B = magneticField(globalPos, Bs.Bq, Bs.Bb);
	B += c2 * magnetization;

	// Linear Zeeman shift
	hamilton.x += abs(INV_SQRT_2 * B.x);
	hamilton.y += abs(INV_SQRT_2 * B.y);
	hamilton.z += abs(B.z);

	size_t idx = zid * dimensions.x * dimensions.y * VALUES_IN_BLOCK + yid * dimensions.x * VALUES_IN_BLOCK + dataXid * VALUES_IN_BLOCK + dualNodeId;
	maxHamlPtr[idx] = max(hamilton.x, max(hamilton.y, hamilton.z));
};

__global__ void density(double* density, PitchedPtr prevStep, uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	char* pPsi = prevStep.ptr + prevStep.slicePitch * zid + prevStep.pitch * yid + sizeof(BlockPsis) * dataXid;
	Complex3Vec psi = ((BlockPsis*)pPsi)->values[dualNodeId];

	size_t idx = VALUES_IN_BLOCK * (zid * dimensions.x * dimensions.y + yid * dimensions.x + dataXid) + dualNodeId;
	density[idx] = (psi.s1 * conj(psi.s1)).x + (psi.s0 * conj(psi.s0)).x + (psi.s_1 * conj(psi.s_1)).x;
}

__global__ void innerProduct(double* result, PitchedPtr pLeft, PitchedPtr pRight, uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	Complex3Vec left = ((BlockPsis*)(pLeft.ptr + pLeft.slicePitch * zid + pLeft.pitch * yid) + dataXid)->values[dualNodeId];
	Complex3Vec right = ((BlockPsis*)(pRight.ptr + pRight.slicePitch * zid + pRight.pitch * yid) + dataXid)->values[dualNodeId];

	size_t idx = VALUES_IN_BLOCK * (zid * dimensions.x * dimensions.y + yid * dimensions.x + dataXid) + dualNodeId;
	result[idx] = (conj(left.s1) * right.s1).x + (conj(left.s0) * right.s0).x + (conj(left.s_1) * right.s_1).x;
}

__global__ void localAvgSpinAndDensity(double* pSpinNorm, double3* pLocalAvgSpin, double* pDensity, PitchedPtr prevStep, uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	char* pPsi = prevStep.ptr + prevStep.slicePitch * zid + prevStep.pitch * yid + sizeof(BlockPsis) * dataXid;
	Complex3Vec psi = ((BlockPsis*)pPsi)->values[dualNodeId];

	double normSq_s1 = psi.s1.x * psi.s1.x + psi.s1.y * psi.s1.y;
	double normSq_s0 = psi.s0.x * psi.s0.x + psi.s0.y * psi.s0.y;
	double normSq_s_1 = psi.s_1.x * psi.s_1.x + psi.s_1.y * psi.s_1.y;

	double density = normSq_s1 + normSq_s0 + normSq_s_1;

	psi.s1 = psi.s1 / sqrt(density);
	psi.s0 = psi.s0 / sqrt(density);
	psi.s_1 = psi.s_1 / sqrt(density);

	double2 temp = SQRT_2 * (conj(psi.s1) * psi.s0 + conj(psi.s0) * psi.s_1);
	double3 localAvgSpin = { temp.x, temp.y, normSq_s1 - normSq_s_1 };

	size_t idx = VALUES_IN_BLOCK * (zid * dimensions.x * dimensions.y + yid * dimensions.x + dataXid) + dualNodeId;

	pSpinNorm[idx] = density * sqrt(localAvgSpin.x * localAvgSpin.x + localAvgSpin.y * localAvgSpin.y + localAvgSpin.z * localAvgSpin.z);
	pLocalAvgSpin[idx] = localAvgSpin;
	pDensity[idx] = density;
}

__global__ void uvTheta(double3* out_u, double3* out_v, double* outTheta, PitchedPtr psiPtr, uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	char* pPsi = psiPtr.ptr + psiPtr.slicePitch * zid + psiPtr.pitch * yid + sizeof(BlockPsis) * dataXid;
	Complex3Vec psi = ((BlockPsis*)pPsi)->values[dualNodeId];

	// a = m + in
	double2 ax = (psi.s_1 - psi.s1) / SQRT_2;
	double2 ay = double2{ 0, -1 } *(psi.s_1 + psi.s1) / SQRT_2;
	double2 az = psi.s0;
	double3 m = double3{ ax.x, ay.x, az.x };
	double3 n = double3{ ax.y, ay.y, az.y };

	double m_dot_n = m.x * n.x + m.y * n.y + m.z * n.z;
	double mNormSqr = m.x * m.x + m.y * m.y + m.z * m.z;
	double nNormSqr = n.x * n.x + n.y * n.y + n.z * n.z;

	double theta = atan2(-2 * m_dot_n, mNormSqr - nNormSqr) / 2;
	if (theta < 0) {
		theta += PI;
	}

	double sinTheta = sin(theta);
	double cosTheta = cos(theta);
	double3 u = double3{ m.x * cosTheta - sinTheta * n.x, m.y * cosTheta - sinTheta * n.y, m.z * cosTheta - sinTheta * n.z };
	double3 v = double3{ m.x * sinTheta + cosTheta * n.x, m.y * sinTheta + cosTheta * n.y, m.z * sinTheta + cosTheta * n.z };
	double uNorm = sqrt(u.x * u.x + u.y * u.y + u.z * u.z);
	double vNorm = sqrt(v.x * v.x + v.y * v.y + v.z * v.z);

	size_t idx = VALUES_IN_BLOCK * (zid * dimensions.x * dimensions.y + yid * dimensions.x + dataXid) + dualNodeId;
	if (uNorm >= vNorm)
	{
		out_u[idx] = u;
		out_v[idx] = v;
	}
	else
	{
		out_u[idx] = v;
		out_v[idx] = u;
	}
	outTheta[idx] = theta;
}

__global__ void integrate(double* dataVec, size_t stride, bool addLast, double dv)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= stride)
	{
		return;
	}

	dataVec[idx] += dataVec[idx + stride];

	if ((idx == (stride - 1)) && addLast)
	{
		dataVec[idx] += dataVec[idx + stride + 1];
	}

	if (stride == 1)
	{
		dataVec[0] *= dv;
	}
}

__global__ void integrateVec(double3* dataVec, size_t stride, bool addLast, double dv)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= stride)
	{
		return;
	}

	dataVec[idx] += dataVec[idx + stride];

	if ((idx == (stride - 1)) && addLast)
	{
		dataVec[idx] += dataVec[idx + stride + 1];
	}

	if (stride == 1)
	{
		dataVec[0] = dv * dataVec[0];
	}
}

__global__ void integrateVecWithDensity(double3* dataVec, double* density, size_t stride, bool addLast, double dv)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= stride)
	{
		return;
	}

	dataVec[idx] = density[idx] * dataVec[idx] + density[idx + stride] * dataVec[idx + stride];

	if ((idx == (stride - 1)) && addLast)
	{
		dataVec[idx] += dataVec[idx + stride + 1];
	}

	if (stride == 1)
	{
		dataVec[0] = dv * dataVec[0];
	}
}


__global__ void reduceMax(double* dataVec, size_t stride, bool addLast)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= stride)
	{
		return;
	}

	dataVec[idx] = max(dataVec[idx], dataVec[idx + stride]);

	if ((idx == (stride - 1)) && addLast)
	{
		dataVec[idx] = max(dataVec[idx], dataVec[idx + stride + 1]);
	}
}

__global__ void normalize(double* density, PitchedPtr psiPtr, uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)
	BlockPsis* blockPsis = (BlockPsis*)(psiPtr.ptr + psiPtr.slicePitch * zid + psiPtr.pitch * yid) + dataXid;
	Complex3Vec psi = blockPsis->values[dualNodeId];
	double sqrtDens = sqrt(density[0]);
	psi.s1 = psi.s1 / sqrtDens;
	psi.s0 = psi.s0 / sqrtDens;
	psi.s_1 = psi.s_1 / sqrtDens;

	blockPsis->values[dualNodeId] = psi;
}

__global__ void polarState(PitchedPtr psi, const uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	BlockPsis* pPsi = (BlockPsis*)(psi.ptr + psi.slicePitch * zid + psi.pitch * yid) + dataXid;

	// Update psi
	size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	Complex3Vec prev = pPsi->values[dualNodeId];

	double normSq_s1 = prev.s1.x * prev.s1.x + prev.s1.y * prev.s1.y;
	double normSq_s0 = prev.s0.x * prev.s0.x + prev.s0.y * prev.s0.y;
	double normSq_s_1 = prev.s_1.x * prev.s_1.x + prev.s_1.y * prev.s_1.y;
	double normSq = normSq_s1 + normSq_s0 + normSq_s_1;

	pPsi->values[dualNodeId].s1 = { 0, 0 };
	pPsi->values[dualNodeId].s0 = { sqrt(normSq), 0 };
	pPsi->values[dualNodeId].s_1 = { 0, 0 };
};

__global__ void ferromagneticState(PitchedPtr psi, const uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	BlockPsis* pPsi = (BlockPsis*)(psi.ptr + psi.slicePitch * zid + psi.pitch * yid) + dataXid;

	// Update psi
	size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	Complex3Vec prev = pPsi->values[dualNodeId];

	double normSq_s1 = prev.s1.x * prev.s1.x + prev.s1.y * prev.s1.y;
	double normSq_s0 = prev.s0.x * prev.s0.x + prev.s0.y * prev.s0.y;
	double normSq_s_1 = prev.s_1.x * prev.s_1.x + prev.s_1.y * prev.s_1.y;
	double normSq = normSq_s1 + normSq_s0 + normSq_s_1;

	pPsi->values[dualNodeId].s1 = { sqrt(normSq), 0 };
	pPsi->values[dualNodeId].s0 = { 0, 0 };
	pPsi->values[dualNodeId].s_1 = { 0, 0 };
};

#if COMPUTE_GROUND_STATE
__global__ void itp_q(PitchedPtr next_q, PitchedPtr prev_q, PitchedPtr psi, int3 * d0, uint3 dimensions, double dt_per_sigma)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;

	size_t dataXid = xid / EDGES_IN_BLOCK; // One thread per every dual edge so EDGES_IN_BLOCK threads per mesh block (on z-axis)
	size_t dualEdgeId = xid % EDGES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	char* pPsi = psi.ptr + psi.slicePitch * zid + psi.pitch * yid + sizeof(BlockPsis) * dataXid;

	Complex3Vec thisPsi = ((BlockPsis*)(pPsi))->values[d0[dualEdgeId].x];
	Complex3Vec otherPsi = ((BlockPsis*)(pPsi + d0[dualEdgeId].y))->values[d0[dualEdgeId].z];
	Complex3Vec d0psi;
	d0psi.s1 = otherPsi.s1 - thisPsi.s1;
	d0psi.s0 = otherPsi.s0 - thisPsi.s0;
	d0psi.s_1 = otherPsi.s_1 - thisPsi.s_1;

	BlockEdges* next = (BlockEdges*)(next_q.ptr + next_q.slicePitch * zid + next_q.pitch * yid) + dataXid;
#if RELATIVISTIC
	BlockEdges* prev = (BlockEdges*)(prev_q.ptr + prev_q.slicePitch * zid + prev_q.pitch * yid) + dataXid;

	Complex3Vec q;
	q.s1  = dt_per_sigma * (-d0psi.s1  + prev->values[dualEdgeId].s1);
	q.s0  = dt_per_sigma * (-d0psi.s0  + prev->values[dualEdgeId].s0);
	q.s_1 = dt_per_sigma * (-d0psi.s_1 + prev->values[dualEdgeId].s_1);

	next->values[dualEdgeId].s1 = prev->values[dualEdgeId].s1 - q.s1;
	next->values[dualEdgeId].s0 = prev->values[dualEdgeId].s0 - q.s0;
	next->values[dualEdgeId].s_1 = prev->values[dualEdgeId].s_1 - q.s_1;
#else
	next->values[dualEdgeId] = d0psi;
#endif
}

__global__ void itp_psi(PitchedPtr HPsiPtr, PitchedPtr nextStep, PitchedPtr prevStep, PitchedPtr qs, const int2 * __restrict__ d1Ptr, const double* __restrict__ hodges, MagFields Bs, const uint3 dimensions, const double block_scale, const double3 p0, const double c0, const double c2, double dt)
{
	const size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	const size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	const size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)
	const size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	// Calculate the pointers for this block
	char* prevPsi = prevStep.ptr + prevStep.slicePitch * zid + prevStep.pitch * yid + sizeof(BlockPsis) * dataXid;
	char* qPtr = qs.ptr + qs.slicePitch * zid + qs.pitch * yid + sizeof(BlockEdges) * dataXid;
	BlockPsis* nextPsi = (BlockPsis*)(nextStep.ptr + nextStep.slicePitch * zid + nextStep.pitch * yid) + dataXid;

	// For computing the energy/chemical potential
	BlockPsis* HPsi = (BlockPsis*)(HPsiPtr.ptr + HPsiPtr.slicePitch * zid + HPsiPtr.pitch * yid) + dataXid;

	// Update psi
	const Complex3Vec prev = ((BlockPsis*)prevPsi)->values[dualNodeId];

	Complex3Vec H;
	H.s1 = { 0, 0 };
	H.s0 = { 0, 0 };
	H.s_1 = { 0, 0 };

	// Add the Laplacian (d1 of d0) to the Hamiltonian
	uint startEdgeId = dualNodeId * FACE_COUNT;
#pragma unroll
	for (int edgeIdOffset = 0; edgeIdOffset < FACE_COUNT; ++edgeIdOffset)
	{
		int edgeId = startEdgeId + edgeIdOffset;
		int2 d1 = d1Ptr[edgeId];
		Complex3Vec d0psi = ((BlockEdges*)(qPtr + d1.x))->values[d1.y];
		const double hodge = hodges[edgeId];

		H.s1 += hodge * d0psi.s1;
		H.s0 += hodge * d0psi.s0;
		H.s_1 += hodge * d0psi.s_1;
	}

	const double normSq_s1 = prev.s1.x * prev.s1.x + prev.s1.y * prev.s1.y;
	const double normSq_s_1 = prev.s_1.x * prev.s_1.x + prev.s_1.y * prev.s_1.y;
	const double normSq = normSq_s1 + (prev.s0.x * prev.s0.x + prev.s0.y * prev.s0.y) + normSq_s_1;

	const double3 localPos = d_localPos[dualNodeId];
	const double3 globalPos = { p0.x + block_scale * (dataXid * BLOCK_WIDTH_X + localPos.x),
		p0.y + block_scale * (yid * BLOCK_WIDTH_Y + localPos.y),
		p0.z + block_scale * (zid * BLOCK_WIDTH_Z + localPos.z) };
	const double totalPot = trap(globalPos) + c0 * normSq;

	H.s1 += totalPot * prev.s1;
	H.s0 += totalPot * prev.s0;
	H.s_1 += totalPot * prev.s_1;

	const double2 magXY = SQRT_2 * (conj(prev.s1) * prev.s0 + conj(prev.s0) * prev.s_1);
	double3 B = c2 * double3{ magXY.x, magXY.y, normSq_s1 - normSq_s_1 };

	// Linear Zeeman shift
	const double2 Bxy = INV_SQRT_2 * double2{ B.x, B.y };
	const double2 BxyConj = conj(Bxy);
	H.s1 += (B.z * prev.s1 + BxyConj * prev.s0);
	H.s0 += (Bxy * prev.s1 + BxyConj * prev.s_1);
	H.s_1 += (Bxy * prev.s0 - B.z * prev.s_1);

	HPsi->values[dualNodeId].s1 = H.s1;
	HPsi->values[dualNodeId].s0 = H.s0;
	HPsi->values[dualNodeId].s_1 = H.s_1;

	nextPsi->values[dualNodeId].s1 = prev.s1 - dt * H.s1;
	nextPsi->values[dualNodeId].s0 = prev.s0 - dt * H.s0;
	nextPsi->values[dualNodeId].s_1 = prev.s_1 - dt * H.s_1;
};
#else
__global__ void forwardEuler(PitchedPtr nextStep, PitchedPtr prevStep, PitchedPtr qs, const int2 * __restrict__ d1Ptr, const double* __restrict__ hodges, MagFields Bs, const uint3 dimensions, const double block_scale, const double3 p0, const double c0, const double c2, double dt)
{
	const size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	const size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	const size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)
	const size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	// Calculate the pointers for this block
	char* prevPsi = prevStep.ptr + prevStep.slicePitch * zid + prevStep.pitch * yid + sizeof(BlockPsis) * dataXid;
	char* qPtr = qs.ptr + qs.slicePitch * zid + qs.pitch * yid + sizeof(BlockEdges) * dataXid;
	BlockPsis* nextPsi = (BlockPsis*)(nextStep.ptr + nextStep.slicePitch * zid + nextStep.pitch * yid) + dataXid;

	// Update psi
	const Complex3Vec prev = ((BlockPsis*)prevPsi)->values[dualNodeId];

	Complex3Vec H;
	H.s1 = { 0, 0 };
	H.s0 = { 0, 0 };
	H.s_1 = { 0, 0 };

	// Add the second spatial exterior derivative (d1 of d0) to the Hamiltonian
	uint startEdgeId = dualNodeId * FACE_COUNT;
#pragma unroll
	for (int edgeIdOffset = 0; edgeIdOffset < FACE_COUNT; ++edgeIdOffset)
	{
		int edgeId = startEdgeId + edgeIdOffset;
		int2 d1 = d1Ptr[edgeId];
		Complex3Vec d0psi = ((BlockEdges*)(qPtr + d1.x))->values[d1.y];
		const double hodge = hodges[edgeId];

		H.s1 += hodge * d0psi.s1;
		H.s0 += hodge * d0psi.s0;
		H.s_1 += hodge * d0psi.s_1;
	}

	const double normSq_s1 = prev.s1.x * prev.s1.x + prev.s1.y * prev.s1.y;
	const double normSq_s_1 = prev.s_1.x * prev.s_1.x + prev.s_1.y * prev.s_1.y;
	const double normSq = normSq_s1 + (prev.s0.x * prev.s0.x + prev.s0.y * prev.s0.y) + normSq_s_1;

	const double3 localPos = d_localPos[dualNodeId];
	const double3 globalPos = { p0.x + block_scale * (dataXid * BLOCK_WIDTH_X + localPos.x),
		p0.y + block_scale * (yid * BLOCK_WIDTH_Y + localPos.y),
		p0.z + block_scale * (zid * BLOCK_WIDTH_Z + localPos.z) };

	double2 totalPot = { trap(globalPos) + c0 * normSq, 0 };

	H.s1 += totalPot * prev.s1;
	H.s0 += totalPot * prev.s0;
	H.s_1 += totalPot * prev.s_1;

	const double2 magXY = SQRT_2 * (conj(prev.s1) * prev.s0 + conj(prev.s0) * prev.s_1);
	double3 B = magneticField(globalPos, Bs.Bq, Bs.Bb);
	B += c2 * double3{ magXY.x, magXY.y, normSq_s1 - normSq_s_1 };

	// Linear Zeeman shift
	double2 Bxy = INV_SQRT_2 * double2{ B.x, B.y };
	double2 BxyConj = conj(Bxy);
	H.s1 += (B.z * prev.s1 + BxyConj * prev.s0);
	H.s0 += (Bxy * prev.s1 + BxyConj * prev.s_1);
	H.s_1 += (Bxy * prev.s0 - B.z * prev.s_1);

	nextPsi->values[dualNodeId].s1 = prev.s1 + dt * double2{ H.s1.y, -H.s1.x };
	nextPsi->values[dualNodeId].s0 = prev.s0 + dt * double2{ H.s0.y, -H.s0.x };
	nextPsi->values[dualNodeId].s_1 = prev.s_1 + dt * double2{ H.s_1.y, -H.s_1.x };
};

__global__ void forwardEuler_q(PitchedPtr next_q, PitchedPtr prev_q, PitchedPtr psi, int3* d0, uint3 dimensions, double dt_per_sigma)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;

	size_t dataXid = xid / EDGES_IN_BLOCK; // One thread per every dual edge so EDGES_IN_BLOCK threads per mesh block (on z-axis)
	size_t dualEdgeId = xid % EDGES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	char* pPsi = psi.ptr + psi.slicePitch * zid + psi.pitch * yid + sizeof(BlockPsis) * dataXid;

	Complex3Vec thisPsi = ((BlockPsis*)(pPsi))->values[d0[dualEdgeId].x];
	Complex3Vec otherPsi = ((BlockPsis*)(pPsi + d0[dualEdgeId].y))->values[d0[dualEdgeId].z];
	Complex3Vec d0psi;
	d0psi.s1 = otherPsi.s1 - thisPsi.s1;
	d0psi.s0 = otherPsi.s0 - thisPsi.s0;
	d0psi.s_1 = otherPsi.s_1 - thisPsi.s_1;

	BlockEdges* next = (BlockEdges*)(next_q.ptr + next_q.slicePitch * zid + next_q.pitch * yid) + dataXid;
#if RELATIVISTIC
	BlockEdges* prev = (BlockEdges*)(prev_q.ptr + prev_q.slicePitch * zid + prev_q.pitch * yid) + dataXid;

	Complex3Vec q;
	q.s1 =  dt_per_sigma * (d0psi.s1  - prev->values[dualEdgeId].s1);
	q.s0 =  dt_per_sigma * (d0psi.s0  - prev->values[dualEdgeId].s0);
	q.s_1 = dt_per_sigma * (d0psi.s_1 - prev->values[dualEdgeId].s_1);

	next->values[dualEdgeId].s1 = prev->values[dualEdgeId].s1  + make_double2(q.s1.y,  -q.s1.x);
	next->values[dualEdgeId].s0 = prev->values[dualEdgeId].s0  + make_double2(q.s0.y,  -q.s0.x);
	next->values[dualEdgeId].s_1 = prev->values[dualEdgeId].s_1 + make_double2(q.s_1.y, -q.s_1.x);
#else
	next->values[dualEdgeId] = d0psi;
#endif
}

__global__ void update_q(PitchedPtr next_q, PitchedPtr prev_q, PitchedPtr psi, int3* d0, uint3 dimensions, double dt_per_sigma)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;

	size_t dataXid = xid / EDGES_IN_BLOCK; // One thread per every dual edge so EDGES_IN_BLOCK threads per mesh block (on z-axis)
	size_t dualEdgeId = xid % EDGES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	char* pPsi = psi.ptr + psi.slicePitch * zid + psi.pitch * yid + sizeof(BlockPsis) * dataXid;

	Complex3Vec thisPsi = ((BlockPsis*)(pPsi))->values[d0[dualEdgeId].x];
	Complex3Vec otherPsi = ((BlockPsis*)(pPsi + d0[dualEdgeId].y))->values[d0[dualEdgeId].z];
	Complex3Vec d0psi;
	d0psi.s1 = otherPsi.s1 - thisPsi.s1;
	d0psi.s0 = otherPsi.s0 - thisPsi.s0;
	d0psi.s_1 = otherPsi.s_1 - thisPsi.s_1;

	BlockEdges* next = (BlockEdges*)(next_q.ptr + next_q.slicePitch * zid + next_q.pitch * yid) + dataXid;
#if RELATIVISTIC
	BlockEdges* prev = (BlockEdges*)(prev_q.ptr + prev_q.slicePitch * zid + prev_q.pitch * yid) + dataXid;

	Complex3Vec q;
	q.s1 =  2 * dt_per_sigma * (d0psi.s1  - prev->values[dualEdgeId].s1);
	q.s0 =  2 * dt_per_sigma * (d0psi.s0  - prev->values[dualEdgeId].s0);
	q.s_1 = 2 * dt_per_sigma * (d0psi.s_1 - prev->values[dualEdgeId].s_1);

	next->values[dualEdgeId].s1 += make_double2( q.s1.y,  -q.s1.x);
	next->values[dualEdgeId].s0 += make_double2( q.s0.y,  -q.s0.x);
	next->values[dualEdgeId].s_1 += make_double2(q.s_1.y, -q.s_1.x);
#else
	next->values[dualEdgeId] = d0psi;
#endif
}

__global__ void update_psi(PitchedPtr nextStep, PitchedPtr prevStep, PitchedPtr qs, const int2* __restrict__ d1Ptr, const double* __restrict__ hodges, MagFields Bs, const uint3 dimensions, const double block_scale, const double3 p0, const double c0, const double c2, double dt)
{
	const size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	const size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	const size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	const size_t dataXid = xid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)
	const size_t dualNodeId = xid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on x-axis)

	// Exit leftover threads
	if (dataXid >= dimensions.x || yid >= dimensions.y || zid >= dimensions.z)
	{
		return;
	}

	// Calculate the pointers for this block
	char* prevPsi = prevStep.ptr + prevStep.slicePitch * zid + prevStep.pitch * yid + sizeof(BlockPsis) * dataXid;
	char* qPtr = qs.ptr + qs.slicePitch * zid + qs.pitch * yid + sizeof(BlockEdges) * dataXid;
	BlockPsis* nextPsi = (BlockPsis*)(nextStep.ptr + nextStep.slicePitch * zid + nextStep.pitch * yid) + dataXid;

	// Update psi
	const Complex3Vec prev = ((BlockPsis*)prevPsi)->values[dualNodeId];

	Complex3Vec H;
	H.s1 = { 0, 0 };
	H.s0 = { 0, 0 };
	H.s_1 = { 0, 0 };

	// Add the the second exterior derivative (d1 of d0) to the Hamiltonian
	uint startEdgeId = dualNodeId * FACE_COUNT;
#pragma unroll
	for (int edgeIdOffset = 0; edgeIdOffset < FACE_COUNT; ++edgeIdOffset)
	{
		int edgeId = startEdgeId + edgeIdOffset;
		int2 d1 = d1Ptr[edgeId];
		Complex3Vec d0psi = ((BlockEdges*)(qPtr + d1.x))->values[d1.y];
		const double hodge = hodges[edgeId];

		H.s1 += hodge * d0psi.s1;
		H.s0 += hodge * d0psi.s0;
		H.s_1 += hodge * d0psi.s_1;
	}

	const double normSq_s1 = prev.s1.x * prev.s1.x + prev.s1.y * prev.s1.y;
	const double normSq_s_1 = prev.s_1.x * prev.s_1.x + prev.s_1.y * prev.s_1.y;
	const double normSq = normSq_s1 + (prev.s0.x * prev.s0.x + prev.s0.y * prev.s0.y) + normSq_s_1;

	const double3 localPos = d_localPos[dualNodeId];
	const double3 globalPos = { p0.x + block_scale * (dataXid * BLOCK_WIDTH_X + localPos.x),
		p0.y + block_scale * (yid * BLOCK_WIDTH_Y + localPos.y),
		p0.z + block_scale * (zid * BLOCK_WIDTH_Z + localPos.z) };

	double2 totalPot = { trap(globalPos) + c0 * normSq, 0 };

	H.s1 += totalPot * prev.s1;
	H.s0 += totalPot * prev.s0;
	H.s_1 += totalPot * prev.s_1;

	const double2 magXY = SQRT_2 * (conj(prev.s1) * prev.s0 + conj(prev.s0) * prev.s_1);
	double3 B = magneticField(globalPos, Bs.Bq, Bs.Bb);
	B += c2 * double3{ magXY.x, magXY.y, normSq_s1 - normSq_s_1 };

	// Linear Zeeman shift
	double2 Bxy = INV_SQRT_2 * double2{ B.x, B.y };
	double2 BxyConj = conj(Bxy);
	H.s1 += (B.z * prev.s1 + BxyConj * prev.s0);
	H.s0 += (Bxy * prev.s1 + BxyConj * prev.s_1);
	H.s_1 += (Bxy * prev.s0 - B.z * prev.s_1);

	nextPsi->values[dualNodeId].s1 += 2 * dt * double2{ H.s1.y, -H.s1.x };
	nextPsi->values[dualNodeId].s0 += 2 * dt * double2{ H.s0.y, -H.s0.x };
	nextPsi->values[dualNodeId].s_1 += 2 * dt * double2{ H.s_1.y, -H.s_1.x };
};
#endif

void normalize_h(dim3 dimGrid, dim3 dimBlock, double* densityPtr, PitchedPtr psi, uint3 dimensions, size_t bodies, double volume)
{
	density << <dimGrid, dimBlock >> > (densityPtr, psi, dimensions);
	int prevStride = bodies;
	while (prevStride > 1)
	{
		int newStride = prevStride / 2;
		integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (densityPtr, newStride, ((newStride * 2) != prevStride), volume);
		prevStride = newStride;
	}

	normalize << < dimGrid, dimBlock >> > (densityPtr, psi, dimensions);
}

void printDensity(dim3 dimGrid, dim3 dimBlock, double* densityPtr, PitchedPtr psi, uint3 dimensions, size_t bodies, double volume)
{
	density << <dimGrid, dimBlock >> > (densityPtr, psi, dimensions);
	int prevStride = bodies;
	while (prevStride > 1)
	{
		int newStride = prevStride / 2;
		integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (densityPtr, newStride, ((newStride * 2) != prevStride), volume);
		prevStride = newStride;
	}
	double hDensity = 0;
	checkCudaErrors(hipMemcpy(&hDensity, densityPtr, sizeof(double), hipMemcpyDeviceToHost));

	std::cout << "Total density: " << hDensity << std::endl;
}

struct SpinMagDens
{
	double spin;
	double3 magnetization;
	double density;
};

SpinMagDens integrateSpinAndDensity(dim3 dimGrid, dim3 dimBlock, double* spinNormPtr, double3* localAvgSpinPtr, double* densityPtr, size_t bodies, double volume)
{
	int prevStride = bodies;
	while (prevStride > 1)
	{
		int newStride = prevStride / 2;
		if (prevStride == bodies)
		{
			integrateVecWithDensity << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (localAvgSpinPtr, densityPtr, newStride, ((newStride * 2) != prevStride), volume);
		}
		else
		{
			integrateVec << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (localAvgSpinPtr, newStride, ((newStride * 2) != prevStride), volume);
		}
		integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (spinNormPtr, newStride, ((newStride * 2) != prevStride), volume);
		integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (densityPtr, newStride, ((newStride * 2) != prevStride), volume);
		prevStride = newStride;
	}
	double3 hMagnetization = { 0, 0, 0 };
	checkCudaErrors(hipMemcpy(&hMagnetization, localAvgSpinPtr, sizeof(double3), hipMemcpyDeviceToHost));

	double hSpinNorm = 0;
	checkCudaErrors(hipMemcpy(&hSpinNorm, spinNormPtr, sizeof(double), hipMemcpyDeviceToHost));

	double hDensity = 0;
	checkCudaErrors(hipMemcpy(&hDensity, densityPtr, sizeof(double), hipMemcpyDeviceToHost));

	return { hSpinNorm, hMagnetization, hDensity };
}

float getMaxHamilton(dim3 dimGrid, dim3 dimBlock, double* maxHamlPtr, PitchedPtr psi, MagFields Bs, uint3 dimensions, size_t bodies, double block_scale, double3 p0)
{
	maxHamilton << <dimGrid, dimBlock >> > (maxHamlPtr, psi, Bs, dimensions, block_scale, p0, c0, c2);
	int prevStride = bodies;
	while (prevStride > 1)
	{
		int newStride = prevStride / 2;
		reduceMax << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (maxHamlPtr, newStride, ((newStride * 2) != prevStride));
		prevStride = newStride;
	}

	double maxHaml = 0;
	checkCudaErrors(hipMemcpy(&maxHaml, maxHamlPtr, sizeof(double), hipMemcpyDeviceToHost));

	return maxHaml;
}

uint integrateInTime(const double block_scale, const Vector3& minp, const Vector3& maxp)
{
	// find dimensions
	const Vector3 domain = maxp - minp;
	const uint xsize = uint(domain.x / (block_scale * BLOCK_WIDTH.x)); // + 1;
	const uint ysize = uint(domain.y / (block_scale * BLOCK_WIDTH.y)); // + 1;
	const uint zsize = uint(domain.z / (block_scale * BLOCK_WIDTH.z)); // + 1;
	const Vector3 p0 = 0.5 * (minp + maxp - block_scale * Vector3(BLOCK_WIDTH.x * xsize, BLOCK_WIDTH.y * ysize, BLOCK_WIDTH.z * zsize));
	const double3 d_p0 = { p0.x, p0.y, p0.z };

	// compute discrete dimensions
	const uint bsize = VALUES_IN_BLOCK; // bpos.size(); // number of values inside a block

	//std::cout << "Dual 0-cells in a replicable structure: " << bsize << std::endl;
	//std::cout << "Replicable structure instances in x: " << xsize << ", y: " << ysize << ", z: " << zsize << std::endl;
	uint64_t bodies = xsize * ysize * zsize * bsize;
	//std::cout << "Dual 0-cells in total: " << bodies << std::endl;

	// Initialize device memory
	size_t dxsize = xsize + 2; // One element buffer to both ends
	size_t dysize = ysize + 2; // One element buffer to both ends
	size_t dzsize = zsize + 2; // One element buffer to both ends
	hipExtent psiExtent = make_hipExtent(dxsize * sizeof(BlockPsis), dysize, dzsize);
	hipExtent edgeExtent = make_hipExtent(dxsize * sizeof(BlockEdges), dysize, dzsize);

	hipPitchedPtr d_cudaEvenPsi;
	hipPitchedPtr d_cudaEvenQ;
	hipPitchedPtr d_cudaOddPsi;
	hipPitchedPtr d_cudaOddQ;
	checkCudaErrors(hipMalloc3D(&d_cudaEvenPsi, psiExtent));
	checkCudaErrors(hipMalloc3D(&d_cudaEvenQ, edgeExtent));
	checkCudaErrors(hipMalloc3D(&d_cudaOddPsi, psiExtent));
	checkCudaErrors(hipMalloc3D(&d_cudaOddQ, edgeExtent));

	// For computing the energy/chemical potential
	hipPitchedPtr d_cudaHPsi;
	checkCudaErrors(hipMalloc3D(&d_cudaHPsi, psiExtent));

	double* d_spinNorm;
	double* d_density;
	double* d_energy;
	double3* d_localAvgSpin;
	double3* d_u;
	double3* d_v;
	double* d_theta;
	checkCudaErrors(hipMalloc(&d_spinNorm, bodies * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_density, bodies * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_energy, bodies * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_localAvgSpin, bodies * sizeof(double3)));
	checkCudaErrors(hipMalloc(&d_u, bodies * sizeof(double3)));
	checkCudaErrors(hipMalloc(&d_v, bodies * sizeof(double3)));
	checkCudaErrors(hipMalloc(&d_theta, bodies * sizeof(double)));

	size_t offset = d_cudaEvenPsi.pitch * dysize + d_cudaEvenPsi.pitch + sizeof(BlockPsis);
	size_t edgeOffset = d_cudaEvenQ.pitch * dysize + d_cudaEvenQ.pitch + sizeof(BlockEdges);
	PitchedPtr d_evenPsi = { (char*)d_cudaEvenPsi.ptr + offset, d_cudaEvenPsi.pitch, d_cudaEvenPsi.pitch * dysize };
	PitchedPtr d_evenQ = { (char*)d_cudaEvenQ.ptr + edgeOffset, d_cudaEvenQ.pitch, d_cudaEvenQ.pitch * dysize };
	PitchedPtr d_oddPsi = { (char*)d_cudaOddPsi.ptr + offset, d_cudaOddPsi.pitch, d_cudaOddPsi.pitch * dysize };
	PitchedPtr d_oddQ = { (char*)d_cudaOddQ.ptr + edgeOffset, d_cudaOddQ.pitch, d_cudaOddQ.pitch * dysize };

	PitchedPtr d_HPsi = { (char*)d_cudaHPsi.ptr + offset, d_cudaHPsi.pitch, d_cudaHPsi.pitch * dysize };

	// find terms for laplacian
	Buffer<int3> d0;
	Buffer<int2> d1;
	Buffer<double> hodges;
	getLaplacian(hodges, d0, d1, sizeof(BlockPsis), d_evenPsi.pitch, d_evenPsi.slicePitch, sizeof(BlockEdges), d_evenQ.pitch, d_evenQ.slicePitch);

	//std::cout << "lapsize = " << lapsize << ", lapfac = " << lapfac << ", lapfac0 = " << lapfac0 << std::endl;

	for (int i = 0; i < hodges.size(); ++i) hodges[i] = -0.5 * hodges[i] / (block_scale * block_scale);

	int3* d_d0;
	checkCudaErrors(hipMalloc(&d_d0, d0.size() * sizeof(int3)));

	int2* d_d1;
	checkCudaErrors(hipMalloc(&d_d1, d1.size() * sizeof(int2)));

	double* d_hodges;
	checkCudaErrors(hipMalloc(&d_hodges, hodges.size() * sizeof(double)));

	// Initialize host memory
	size_t hostSize = dxsize * dysize * dzsize;
	BlockPsis* h_evenPsi;
	BlockPsis* h_oddPsi;
	BlockPsis* h_evenQ;
	BlockPsis* h_oddQ;
	checkCudaErrors(hipHostMalloc(&h_evenPsi, hostSize * sizeof(BlockPsis)));
	checkCudaErrors(hipHostMalloc(&h_oddPsi, hostSize * sizeof(BlockPsis)));
	checkCudaErrors(hipHostMalloc(&h_evenQ, hostSize * sizeof(BlockEdges)));
	checkCudaErrors(hipHostMalloc(&h_oddQ, hostSize * sizeof(BlockEdges)));
	memset(h_evenPsi, 0, hostSize * sizeof(BlockPsis));
	memset(h_oddPsi, 0, hostSize * sizeof(BlockPsis));
	memset(h_evenQ, 0, hostSize * sizeof(BlockEdges));
	memset(h_oddQ, 0, hostSize * sizeof(BlockEdges));

	double* h_density;
	double3* h_u;
	double* h_theta;
	double3* h_localAvgSpin;
	checkCudaErrors(hipHostMalloc(&h_density, bodies * sizeof(double)));
	checkCudaErrors(hipHostMalloc(&h_u, bodies * sizeof(double3)));
	checkCudaErrors(hipHostMalloc(&h_theta, bodies * sizeof(double)));
	checkCudaErrors(hipHostMalloc(&h_localAvgSpin, bodies * sizeof(double3)));

#if COMPUTE_GROUND_STATE
	// Initialize discrete field
	std::ifstream fs(GROUND_STATE_PSI_FILENAME, std::ios::binary | std::ios::in);
	bool continueFromEarlier = (fs.fail() == 0);
	if (continueFromEarlier)
	{
		std::cout << "Initialized ground state psi from file." << std::endl;

		fs.read((char*)&h_evenPsi[0], hostSize * sizeof(BlockPsis));
		fs.close();

		std::ifstream fs_q(GROUND_STATE_Q_FILENAME, std::ios::binary | std::ios::in);
		if (fs.fail() == 0)
		{
			std::cout << "Initialized ground state q from file." << std::endl;

			fs.read((char*)&h_evenQ[0], hostSize * sizeof(BlockEdges));
			fs.close();
		}
		else
		{
			std::cout << "Failed to open the ground state q file." << std::endl;
		}
	}
	else
	{
		std::cout << "Initialized ground state with random noise." << std::endl;

		std::default_random_engine generator;
		std::normal_distribution<double> distribution(0.0, 1.0);
		for (uint k = 0; k < zsize; k++)
		{
			for (uint j = 0; j < ysize; j++)
			{
				for (uint i = 0; i < xsize; i++)
				{
					for (uint l = 0; l < bsize; l++)
					{
						const uint dstI = (k + 1) * dxsize * dysize + (j + 1) * dxsize + (i + 1);
						const double2 s1{ distribution(generator), distribution(generator) };
						const double2 s0{ distribution(generator), distribution(generator) };
						const double2 s_1{ distribution(generator), distribution(generator) };
						h_evenPsi[dstI].values[l].s1 = s1;
						h_evenPsi[dstI].values[l].s0 = s0;
						h_evenPsi[dstI].values[l].s_1 = s_1;
					}
				}
			}
		}
	}

	bool loadGroundState = false;
	bool doForward = false;
#else
	bool loadGroundState = (t == 0);
	std::string psi_filename = loadGroundState ? GROUND_STATE_PSI_FILENAME : toString(t) + ".dat";
	std::ifstream psi_fs(psi_filename, std::ios::binary | std::ios::in);
	if (psi_fs.fail() != 0)
	{
		std::cout << "Failed to open file " << psi_filename << std::endl;
		return 1;
	}
	else
	{
		std::cout << "Loading ground state psi from " << psi_filename << "..." << std::endl;
	}
	psi_fs.read((char*)&h_oddPsi[0], hostSize * sizeof(BlockPsis));
	psi_fs.close();

#if RELATIVISTIC
	std::string q_filename = loadGroundState ? GROUND_STATE_Q_FILENAME : toString(t) + ".dat";
	std::ifstream q_fs(q_filename, std::ios::binary | std::ios::in);
	if (q_fs.fail() != 0)
	{
		std::cout << "Failed to open file " << q_filename << std::endl;
		return 1;
	}
	else
	{
		std::cout << "Loading ground state q from " << q_filename << "..." << std::endl;
	}
	q_fs.read((char*)&h_oddQ[0], hostSize * sizeof(BlockEdges));
	q_fs.close();
#endif

	bool doForward = true;
	std::string evenFilename = "even_" + toString(t) + ".dat";
	std::ifstream evenFs(evenFilename, std::ios::binary | std::ios::in);
	if (evenFs.fail() == 0)
	{
		evenFs.read((char*)&h_evenPsi[0], hostSize * sizeof(BlockPsis));
		evenFs.close();
		doForward = false;
		std::cout << "Loaded even time step from file" << std::endl;
	}

#endif

	hipPitchedPtr h_cudaEvenPsi = { 0 };
	hipPitchedPtr h_cudaOddPsi = { 0 };
	hipPitchedPtr h_cudaEvenQ = { 0 };
	hipPitchedPtr h_cudaOddQ = { 0 };

	h_cudaEvenPsi.ptr = h_evenPsi;
	h_cudaEvenPsi.pitch = dxsize * sizeof(BlockPsis);
	h_cudaEvenPsi.xsize = d_cudaEvenPsi.xsize;
	h_cudaEvenPsi.ysize = d_cudaEvenPsi.ysize;

	h_cudaOddPsi.ptr = h_oddPsi;
	h_cudaOddPsi.pitch = dxsize * sizeof(BlockPsis);
	h_cudaOddPsi.xsize = d_cudaOddPsi.xsize;
	h_cudaOddPsi.ysize = d_cudaOddPsi.ysize;

	h_cudaEvenQ.ptr = h_evenQ;
	h_cudaEvenQ.pitch = dxsize * sizeof(BlockEdges);
	h_cudaEvenQ.xsize = d_cudaEvenQ.xsize;
	h_cudaEvenQ.ysize = d_cudaEvenQ.ysize;

	h_cudaOddQ.ptr = h_oddQ;
	h_cudaOddQ.pitch = dxsize * sizeof(BlockEdges);
	h_cudaOddQ.xsize = d_cudaOddQ.xsize;
	h_cudaOddQ.ysize = d_cudaOddQ.ysize;

	// Copy from host memory to device memory
	hipMemcpy3DParms evenPsiParams = { 0 };
	hipMemcpy3DParms oddPsiParams = { 0 };

	evenPsiParams.srcPtr = h_cudaEvenPsi;
	evenPsiParams.dstPtr = d_cudaEvenPsi;
	evenPsiParams.extent = psiExtent;
	evenPsiParams.kind = hipMemcpyHostToDevice;

	oddPsiParams.srcPtr = h_cudaOddPsi;
	oddPsiParams.dstPtr = d_cudaOddPsi;
	oddPsiParams.extent = psiExtent;
	oddPsiParams.kind = hipMemcpyHostToDevice;

	hipMemcpy3DParms evenQParams = { 0 };
	hipMemcpy3DParms oddQParams = { 0 };
	
	evenQParams.srcPtr = h_cudaEvenQ;
	evenQParams.dstPtr = d_cudaEvenQ;
	evenQParams.extent = edgeExtent;
	evenQParams.kind = hipMemcpyHostToDevice;
	
	oddQParams.srcPtr = h_cudaOddQ;
	oddQParams.dstPtr = d_cudaOddQ;
	oddQParams.extent = edgeExtent;
	oddQParams.kind = hipMemcpyHostToDevice;

	checkCudaErrors(hipMemcpy3D(&evenPsiParams));
	checkCudaErrors(hipMemcpy3D(&oddPsiParams));
	checkCudaErrors(hipMemcpy3D(&evenQParams));
	checkCudaErrors(hipMemcpy3D(&oddQParams));
	checkCudaErrors(hipMemcpy(d_d0, &d0[0], d0.size() * sizeof(int3), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_d1, &d1[0], d1.size() * sizeof(int2), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_hodges, &hodges[0], hodges.size() * sizeof(double), hipMemcpyHostToDevice));

	// Clear host memory after data has been copied to devices
	hipDeviceSynchronize();
	d0.clear();
	d1.clear();
	hodges.clear();
#if !(SAVE_PICTURE)
	hipHostFree(h_evenPsi);
	hipHostFree(h_oddPsi);
#endif
	hipMemcpy3DParms evenPsiBackParams = { 0 };
	evenPsiBackParams.srcPtr = d_cudaEvenPsi;
	evenPsiBackParams.dstPtr = h_cudaEvenPsi;
	evenPsiBackParams.extent = psiExtent;
	evenPsiBackParams.kind = hipMemcpyDeviceToHost;

	hipMemcpy3DParms oddPsiBackParams = { 0 };
	oddPsiBackParams.srcPtr = d_cudaOddPsi;
	oddPsiBackParams.dstPtr = h_cudaOddPsi;
	oddPsiBackParams.extent = psiExtent;
	oddPsiBackParams.kind = hipMemcpyDeviceToHost;

	hipMemcpy3DParms evenQBackParams = { 0 };
	evenQBackParams.srcPtr = d_cudaEvenQ;
	evenQBackParams.dstPtr = h_cudaEvenQ;
	evenQBackParams.extent = edgeExtent;
	evenQBackParams.kind = hipMemcpyDeviceToHost;

	hipMemcpy3DParms oddQBackParams = { 0 };
	oddQBackParams.srcPtr = d_cudaOddQ;
	oddQBackParams.dstPtr = h_cudaOddQ;
	oddQBackParams.extent = edgeExtent;
	oddQBackParams.kind = hipMemcpyDeviceToHost;

	// Integrate in time
	uint3 dimensions = make_uint3(xsize, ysize, zsize);
	dim3 psiDimBlock(THREAD_BLOCK_X * VALUES_IN_BLOCK, THREAD_BLOCK_Y, THREAD_BLOCK_Z);
	dim3 edgeDimBlock(THREAD_BLOCK_X * EDGES_IN_BLOCK, THREAD_BLOCK_Y, THREAD_BLOCK_Z);
	dim3 dimGrid((xsize + THREAD_BLOCK_X - 1) / THREAD_BLOCK_X,
		(ysize + THREAD_BLOCK_Y - 1) / THREAD_BLOCK_Y,
		((zsize + THREAD_BLOCK_Z - 1) / THREAD_BLOCK_Z));

	Signal signal;
	MagFields Bs{};

	const double volume = block_scale * block_scale * block_scale * VOLUME;

	if (loadGroundState)
	{
		switch (initPhase)
		{
		case Phase::Polar:
			std::cout << "Transform ground state to polar phase" << std::endl;
			polarState << <dimGrid, psiDimBlock >> > (d_oddPsi, dimensions);
			break;
		case Phase::Ferromagnetic:
			std::cout << "Transform ground state to ferromagnetic phase" << std::endl;
			ferromagneticState << <dimGrid, psiDimBlock >> > (d_oddPsi, dimensions);
			break;
		default:
			break;
		}

		printDensity(dimGrid, psiDimBlock, d_density, d_oddPsi, dimensions, bodies, volume);
	}

#if !COMPUTE_GROUND_STATE
	// Take one forward Euler step if starting from the ground state or time step changed
	if (doForward)
	{
		std::cout << "No even time step file found. Doing one forward step." << std::endl;

		signal = getSignal(t);
		Bs.Bq = BqScale * signal.Bq;
		Bs.Bb = BzScale * signal.Bb;
		Bs.BqQuad = BqQuadScale * signal.Bq;
		Bs.BbQuad = BzQuadScale * signal.Bb;
#if RELATIVISTIC
		forwardEuler << <dimGrid, psiDimBlock >> > (d_evenPsi, d_oddPsi, d_oddQ, d_d1, d_hodges, Bs, dimensions, block_scale, d_p0, c0, c2, dt);
		forwardEuler_q << <dimGrid, edgeDimBlock >> > (d_evenQ, d_oddQ, d_oddPsi, d_d0, dimensions, dt_per_sigma);
#else
		forwardEuler_q << <dimGrid, edgeDimBlock >> > (d_oddQ, d_oddQ, d_oddPsi, d_d0, dimensions, dt_per_sigma);
		forwardEuler << <dimGrid, psiDimBlock >> > (d_evenPsi, d_oddPsi, d_oddQ, d_d1, d_hodges, Bs, dimensions, block_scale, d_p0, c0, c2, dt);
		forwardEuler_q << <dimGrid, edgeDimBlock >> > (d_evenQ, d_evenQ, d_evenPsi, d_d0, dimensions, dt_per_sigma);
#endif
	}
	else
#endif
	{
		std::cout << "Skipping the forward step." << std::endl;
	}

#if COMPUTE_GROUND_STATE
	std::string folder = "gs_dens_profiles";
	std::string createResultsDirCommand = "mkdir " + folder;
	system(createResultsDirCommand.c_str());

	uint iter = 0;
	
	if (!continueFromEarlier)
	{
		normalize_h(dimGrid, psiDimBlock, d_density, d_evenPsi, dimensions, bodies, volume);
		normalize_h(dimGrid, psiDimBlock, d_density, d_oddPsi, dimensions, bodies, volume);
		itp_q << <dimGrid, edgeDimBlock >> > (d_evenQ, d_evenQ, d_evenPsi, d_d0, dimensions, dt_per_sigma);
		itp_q << <dimGrid, edgeDimBlock >> > (d_oddQ, d_oddQ, d_oddPsi, d_d0, dimensions, dt_per_sigma);
	}

	while (true)
	{
		if ((iter % 1000) == 0) std::cout << "Iteration " << iter << std::endl;
#if SAVE_PICTURE
		if ((iter % 1000) == 0)
		{
			checkCudaErrors(hipMemcpy3D(&evenPsiBackParams));
			drawDensity(h_evenPsi, dxsize, dysize, dzsize, iter, folder);
			printDensity(dimGrid, psiDimBlock, d_density, d_evenPsi, dimensions, bodies, volume);

			// Compute energy/chemical potential
			innerProduct << <dimGrid, psiDimBlock >> > (d_energy, d_evenPsi, d_HPsi, dimensions);
			int prevStride = bodies;
			while (prevStride > 1)
			{
				int newStride = prevStride / 2;
				integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (d_energy, newStride, ((newStride * 2) != prevStride), volume);
				prevStride = newStride;
			}
			double hEnergy = 0;
			checkCudaErrors(hipMemcpy(&hEnergy, d_energy, sizeof(double), hipMemcpyDeviceToHost));
			std::cout << "Energy: " << hEnergy << std::endl;
		}
#endif
		if (iter == 500000)
		{
			// Psi
			checkCudaErrors(hipMemcpy3D(&evenPsiBackParams));
			std::ofstream fs_psi(GROUND_STATE_PSI_FILENAME, std::ios::binary | std::ios_base::trunc);
			if (fs_psi.fail() != 0) return 1;
			fs_psi.write((char*)&h_evenPsi[0], hostSize * sizeof(BlockPsis));
			fs_psi.close();

			// Q
			checkCudaErrors(hipMemcpy3D(&evenQBackParams));
			std::ofstream fs_q(GROUND_STATE_Q_FILENAME, std::ios::binary | std::ios_base::trunc);
			if (fs_q.fail() != 0) return 1;
			fs_q.write((char*)&h_evenQ[0], hostSize * sizeof(BlockEdges));
			fs_q.close();

			return 0;
		}
#if RELATIVISTIC
		// Take an imaginary time step
		itp_q << <dimGrid, edgeDimBlock >> > (d_oddQ, d_evenQ, d_evenPsi, d_d0, dimensions, dt_per_sigma);
		itp_psi << <dimGrid, psiDimBlock >> > (d_HPsi, d_oddPsi, d_evenPsi, d_evenQ, d_d1, d_hodges, Bs, dimensions, block_scale, d_p0, c0, c2, dt);
		// Normalize
		normalize_h(dimGrid, psiDimBlock, d_density, d_oddPsi, dimensions, bodies, volume);

		// Take an imaginary time step
		itp_q << <dimGrid, edgeDimBlock >> > (d_evenQ, d_oddQ, d_oddPsi, d_d0, dimensions, dt_per_sigma);
		itp_psi << <dimGrid, psiDimBlock >> > (d_HPsi, d_evenPsi, d_oddPsi, d_oddQ, d_d1, d_hodges, Bs, dimensions, block_scale, d_p0, c0, c2, dt);
		// Normalize
		normalize_h(dimGrid, psiDimBlock, d_density, d_evenPsi, dimensions, bodies, volume);
#endif

		iter++;
	}

#else
	std::string times = std::string("times = [times");
	std::string bqString = std::string("Bq = [Bq");
	std::string bzString = std::string("Bz = [Bz");
	std::string spinString = std::string("Spin = [Spin");
	std::string magX = std::string("mag_x = [mag_x");
	std::string magY = std::string("mag_y = [mag_y");
	std::string magZ = std::string("mag_z = [mag_z");
	std::string densityStr = std::string("norm = [norm");

	int lastSaveTime = 0;

#if RELATIVISTIC
	std::string folder = "hyperbolic";
#else
	std::string folder = "parabolic";
#endif

	std::string createResultsDirCommand = "mkdir " + folder;
	system(createResultsDirCommand.c_str());

	while (t < END_TIME)
	{
#if SAVE_PICTURE
		// Copy back from device memory to host memory
		checkCudaErrors(hipMemcpy3D(&oddPsiBackParams));

		// Measure wall clock time
		static auto prevTime = std::chrono::high_resolution_clock::now();
		auto duration = std::chrono::high_resolution_clock::now() - prevTime;
		std::cout << "Simulation time: " << t << " ms. Real time from previous save: " << duration.count() * 1e-9 << " s." << std::endl;
		prevTime = std::chrono::high_resolution_clock::now();

#if RELATIVISTIC
		drawDensity(h_oddPsi, dxsize, dysize, dzsize, t, folder);
		//drawDensityRI(h_oddPsi, dxsize, dysize, dzsize, t, "energy_test");
#else
		drawDensity(h_oddPsi, dxsize, dysize, dzsize, t, folder);
#endif

		//uvTheta << <dimGrid, dimBlock >> > (d_u, d_v, d_theta, d_oddPsi, dimensions);
		//hipMemcpy(h_u, d_u, bodies * sizeof(double3), hipMemcpyDeviceToHost);
		//hipMemcpy(h_theta, d_theta, bodies * sizeof(double), hipMemcpyDeviceToHost);
		//drawUtheta(h_u, h_theta, xsize, ysize, zsize, t - 202.03);
		//
		//ferromagneticDomain << <dimGrid, dimBlock >> > (d_ferroDom, d_oddPsi, dimensions);
		//hipMemcpy(h_ferroDom, d_ferroDom, bodies * sizeof(double), hipMemcpyDeviceToHost);
		//drawFerroDom(h_ferroDom, xsize, ysize, zsize, t - 202.03);
#endif
		const uint centerIdx = 57 * dxsize * dysize + 57 * dxsize + 57;
		double2 temp = h_oddPsi[centerIdx].values[5].s0;
		double startPhase = atan2(temp.y, temp.x);
		double phaseTime = 0;

		// integrate one iteration
		for (uint step = 0; step < IMAGE_SAVE_FREQUENCY; step++)
		{
			// update odd values (imaginary terms)
			phaseTime += dt;
			t += dt / omega_r * 1e3; // [ms]
			signal = getSignal(t);
			Bs.Bq = BqScale * signal.Bq;
			Bs.Bb = BzScale * signal.Bb;
			Bs.BqQuad = BqQuadScale * signal.Bq;
			Bs.BbQuad = BzQuadScale * signal.Bb;
			update_psi << <dimGrid, psiDimBlock >> > (d_oddPsi, d_evenPsi, d_evenQ, d_d1, d_hodges, Bs, dimensions, block_scale, d_p0, c0, c2, dt);
#if RELATIVISTIC
			update_q << <dimGrid, edgeDimBlock >> > (d_oddQ, d_evenQ, d_evenPsi, d_d0, dimensions, dt_per_sigma);
#else
			update_q << <dimGrid, edgeDimBlock >> > (d_oddQ, d_oddQ, d_oddPsi, d_d0, dimensions, dt_per_sigma);
#endif

			// update even values (real terms)
			phaseTime += dt;
			t += dt / omega_r * 1e3; // [ms]
			signal = getSignal(t);
			Bs.Bq = BqScale * signal.Bq;
			Bs.Bb = BzScale * signal.Bb;
			Bs.BqQuad = BqQuadScale * signal.Bq;
			Bs.BbQuad = BzQuadScale * signal.Bb;
			update_psi << <dimGrid, psiDimBlock >> > (d_evenPsi, d_oddPsi, d_oddQ, d_d1, d_hodges, Bs, dimensions, block_scale, d_p0, c0, c2, dt);
#if RELATIVISTIC
			update_q << <dimGrid, edgeDimBlock >> > (d_evenQ, d_oddQ, d_oddPsi, d_d0, dimensions, dt_per_sigma);
#else
			update_q << <dimGrid, edgeDimBlock >> > (d_evenQ, d_evenQ, d_evenPsi, d_d0, dimensions, dt_per_sigma);
#endif
		}
		// Copy back from device memory to host memory
		checkCudaErrors(hipMemcpy3D(&oddPsiBackParams));

		temp = h_oddPsi[centerIdx].values[5].s0;
		double endPhase = atan2(temp.y, temp.x);
		double phaseDiff = endPhase - startPhase;
		std::cout << "Energy was " << phaseDiff / phaseTime << std::endl;

#if SAVE_STATES
		// Copy back from device memory to host memory
		checkCudaErrors(hipMemcpy3D(&oddPsiBackParams));

		localAvgSpinAndDensity << <dimGrid, psiDimBlock >> > (d_spinNorm, d_localAvgSpin, d_density, d_oddPsi, dimensions);
		hipMemcpy(h_localAvgSpin, d_localAvgSpin, bodies * sizeof(double3), hipMemcpyDeviceToHost);
		uvTheta << <dimGrid, psiDimBlock >> > (d_u, d_v, d_theta, d_oddPsi, dimensions);
		hipMemcpy(h_u, d_u, bodies * sizeof(double3), hipMemcpyDeviceToHost);
		hipMemcpy(h_theta, d_theta, bodies * sizeof(double), hipMemcpyDeviceToHost);
		saveVolume(SAVE_FILE_PREFIX, h_oddPsi, h_localAvgSpin, h_u, h_theta, bsize, dxsize, dysize, dzsize, 0, block_scale, d_p0, t - 202.03);

		SpinMagDens spinMagDens = integrateSpinAndDensity(dimGrid, psiDimBlock, d_spinNorm, d_localAvgSpin, d_density, bodies, volume);
		times += ", " + toString(t);
		bqString += ", " + toString(Bs.Bq);
		bzString += ", " + toString(Bs.Bz);
		spinString += ", " + toString(spinMagDens.spin);
		magX += ", " + toString(spinMagDens.magnetization.x);
		magY += ", " + toString(spinMagDens.magnetization.y);
		magZ += ", " + toString(spinMagDens.magnetization.z);
		densityStr += ", " + toString(spinMagDens.density);

		if (((int(t) % STATE_SAVE_INTERVAL) == 0) && (int(t) != lastSaveTime))
		{
			times += "];";
			bqString += "];";
			bzString += "];";
			spinString += "];";
			magX += "];";
			magY += "];";
			magZ += "];";
			densityStr += "];";

			Text textFile;
			textFile << times << std::endl;
			textFile << bqString << std::endl;
			textFile << bzString << std::endl;
			textFile << spinString << std::endl;
			textFile << magX << std::endl;
			textFile << magY << std::endl;
			textFile << magZ << std::endl;
			textFile << densityStr << std::endl;
			textFile.save(SAVE_FILE_PREFIX + toString(t) + ".m");

			std::ofstream oddFs(SAVE_FILE_PREFIX + toString(t) + ".dat", std::ios::binary | std::ios_base::trunc);
			if (oddFs.fail() != 0) return 1;
			oddFs.write((char*)&h_oddPsi[0], hostSize * sizeof(BlockPsis));
			oddFs.close();

			checkCudaErrors(hipMemcpy3D(&evenPsiBackParams));
			std::ofstream evenFs(SAVE_FILE_PREFIX + "even_" + toString(t) + ".dat", std::ios::binary | std::ios_base::trunc);
			if (evenFs.fail() != 0) return 1;
			evenFs.write((char*)&h_evenPsi[0], hostSize * sizeof(BlockPsis));
			evenFs.close();

			std::cout << "Saved the state!" << std::endl;

			times = std::string("times = [times");
			bqString = std::string("Bq = [Bq");
			bzString = std::string("Bz = [Bz");
			spinString = std::string("Spin = [Spin");
			magX = std::string("mag_x = [mag_x");
			magY = std::string("mag_y = [mag_y");
			magZ = std::string("mag_z = [mag_z");
			densityStr = std::string("norm = [norm");

			lastSaveTime = int(t);
		}
#endif
	}
#endif

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernels (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	return 0;
}

void readConfFile()
{
	std::ifstream file;
	file.open("conf.conf", std::ios::in);
	if (file.is_open())
	{
		std::string line;
		while (std::getline(file, line))
		{
			if (size_t pos = line.find("t0") != std::string::npos)
			{
				t = std::stod(line.substr(pos + 2));
			}
			else if (size_t pos = line.find("end") != std::string::npos)
			{
				END_TIME = std::stod(line.substr(pos + 3));
			}
			else if (size_t pos = line.find("dt") != std::string::npos)
			{
				dt = std::stod(line.substr(pos + 2));
				IMAGE_SAVE_FREQUENCY = uint(IMAGE_SAVE_INTERVAL * 0.5 / 1e3 * omega_r / dt) + 1;
				dt_per_sigma = dt / sigma;
			}
			else if (size_t pos = line.find("sigma") != std::string::npos)
			{
				sigma = std::stod(line.substr(pos + 5));
				dt_per_sigma = dt / sigma;
			}
		}
	}
}

int main(int argc, char** argv)
{
	readConfFile();

	const double blockScale = DOMAIN_SIZE_X / REPLICABLE_STRUCTURE_COUNT_X / BLOCK_WIDTH_X;

	std::cout << "Start simulating from t = " << t << " ms, with a time step size of " << dt << "." << std::endl;
	std::cout << "The simulation will end at " << END_TIME << " ms." << std::endl;
	//std::cout << "Block scale = " << blockScale << std::endl;
	//std::cout << "Dual edge length = " << DUAL_EDGE_LENGTH * blockScale << std::endl;
	std::cout << "Relativistic sigma = " << sigma << std::endl;

	// integrate in time using DEC
	auto domainMin = Vector3(-DOMAIN_SIZE_X * 0.5, -DOMAIN_SIZE_Y * 0.5, -DOMAIN_SIZE_Z * 0.5);
	auto domainMax = Vector3(DOMAIN_SIZE_X * 0.5, DOMAIN_SIZE_Y * 0.5, DOMAIN_SIZE_Z * 0.5);
	integrateInTime(blockScale, domainMin, domainMax);

	return 0;
}
