#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "VortexState.hpp"
#include <Output/Picture.hpp>
#include <Output/Text.hpp>
#include <Types/Complex.hpp>
#include <Types/Random.hpp>
#include <Mesh/DelaunayMesh.hpp>
#include <iostream>
#include <sstream>

#include <mesh.h>

ddouble RATIO = 1.0;
ddouble KAPPA = 10;
ddouble G = 300;

#define LOAD_STATE_FROM_DISK 1
#define SAVE_PICTURE 1
#define SAVE_VOLUME 0

#define THREAD_BLOCK_X 8
#define THREAD_BLOCK_Y 8
#define THREAD_BLOCK_Z 1

ddouble potentialRZ(const ddouble r, const ddouble z)
{
	return 0.5 * (r * r + RATIO * RATIO * z * z);
}

ddouble potentialV3(const Vector3& p)
{
	return 0.5 * (p.x * p.x + p.y * p.y + RATIO * RATIO * p.z * p.z);
}

bool saveVolumeMap(const std::string& path, const Buffer<ushort>& vol, const uint xsize, const uint ysize, const uint zsize, const Vector3& h)
{
	Text rawpath;
	rawpath << path << ".raw";

	// save raw
	std::ofstream fs(rawpath.str().c_str(), std::ios_base::binary | std::ios::trunc);
	if (fs.fail()) return false;
	fs.write((char*)&vol[0], 2 * xsize * ysize * zsize);
	fs.close();

	// save header
	Text text;

	text << "ObjectType              = Image" << std::endl;
	text << "NDims                   = 3" << std::endl;
	text << "BinaryData              = True" << std::endl;
	text << "CompressedData          = False" << std::endl;
	text << "BinaryDataByteOrderMSB  = False" << std::endl;
	text << "TransformMatrix         = 1 0 0 0 1 0 0 0 1" << std::endl;
	text << "Offset                  = " << -0.5 * xsize * h.x << " " << -0.5 * ysize * h.y << " " << -0.5 * zsize * h.z << std::endl;
	text << "CenterOfRotation        = 0 0 0" << std::endl;
	text << "DimSize                 = " << xsize << " " << ysize << " " << zsize << std::endl;
	text << "ElementSpacing          = " << h.x << " " << h.y << " " << h.z << std::endl;
	text << "ElementNumberOfChannels = 1" << std::endl;
	text << "ElementType             = MET_USHORT" << std::endl;
	text << "ElementDataFile         = " << rawpath.str() << std::endl;
	text.save(path);
	return true;
}

struct BlockPsis
{
	double values[VALUES_IN_BLOCK];
};

struct BlockEdges
{
	double values[EDGES_IN_BLOCK];
};

struct BlockPots
{
	double values[VALUES_IN_BLOCK];
};

struct PitchedPtr
{
	char* ptr;
	size_t pitch;
	size_t slicePitch;
};

__global__ void update1forms(PitchedPtr nextEdge, PitchedPtr prevEdge, PitchedPtr psis, int2* lapInd, double* hodges, double g, uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;

	size_t dataZid = zid / EDGES_IN_BLOCK; // One thread per every dual edge so EDGES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (xid >= dimensions.x || yid >= dimensions.y || dataZid >= dimensions.z)
	{
		return;
	}

	char* pPsi = psis.ptr + psis.slicePitch * dataZid + psis.pitch * yid + sizeof(BlockPsis) * xid;
	double psi = ((BlockPsis*)pPsi)->values[0];
	
	double e0 = hodges[3] * (((BlockPsis*)(pPsi + lapInd[3].x))->values[lapInd[3].y] - psi);
	double e1 = hodges[3] * (((BlockPsis*)(pPsi + lapInd[4].x))->values[lapInd[4].y] - psi);
	double e2 = hodges[3] * (((BlockPsis*)(pPsi + lapInd[5].x))->values[lapInd[5].y] - psi);

	BlockEdges* nextPsi = (BlockEdges*)(nextEdge.ptr + nextEdge.slicePitch * dataZid + nextEdge.pitch * yid) + xid;
}

__global__ void update(PitchedPtr nextStep, PitchedPtr prevStep, PitchedPtr potentials, int2* lapInd, double* hodges, double g, uint3 dimensions, double sign)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;

	size_t dataZid = zid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (xid >= dimensions.x || yid >= dimensions.y || dataZid >= dimensions.z)
	{
		return;
	}

	// Calculate the pointers for this block
	char* prevPsi = prevStep.ptr + prevStep.slicePitch * dataZid + prevStep.pitch * yid + sizeof(BlockPsis) * xid;
	BlockPsis* nextPsi = (BlockPsis*)(nextStep.ptr + nextStep.slicePitch * dataZid + nextStep.pitch * yid) + xid;
	BlockPots* pot = (BlockPots*)(potentials.ptr + potentials.slicePitch * dataZid + potentials.pitch * yid) + xid;

	// Update psi
	size_t dualNodeId = zid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)
	double prev = ((BlockPsis*)prevPsi)->values[dualNodeId];

	uint primaryFace = dualNodeId * FACE_COUNT;
	double sum = 0;
#pragma unroll
	for (int i = 0; i < FACE_COUNT; ++i)
		sum += hodges[primaryFace] * (((BlockPsis*)(prevPsi + lapInd[primaryFace].x))->values[lapInd[primaryFace++].y] - prev);

	double next = nextPsi->values[dualNodeId];

	double normsq = prev * prev + next * next;
	sum += (pot->values[dualNodeId] + g * normsq) * prev;

	nextPsi->values[dualNodeId] = next + sign * sum;
};

uint integrateInTime(const VortexState& state, const ddouble block_scale, const Vector3& minp, const Vector3& maxp, const ddouble iteration_period, const uint number_of_iterations)
{
	uint i, j, k, l;

	// find dimensions
	const Vector3 domain = maxp - minp;
	const uint xsize = uint(domain.x / (block_scale * BLOCK_WIDTH.x)) + 1;
	const uint ysize = uint(domain.y / (block_scale * BLOCK_WIDTH.y)) + 1;
	const uint zsize = uint(domain.z / (block_scale * BLOCK_WIDTH.z)) + 1;
	const Vector3 p0 = 0.5 * (minp + maxp - block_scale * Vector3(BLOCK_WIDTH.x * xsize, BLOCK_WIDTH.y * ysize, BLOCK_WIDTH.z * zsize));

	//std::cout << xsize << ", " << ysize << ", " << zsize << std::endl;

	// find relative circumcenters for each body element
	Buffer<Vector3> bpos;
	getPositions(bpos);

	// compute discrete dimensions
	const uint bsize = bpos.size(); // number of values inside a block
	const uint bxsize = (xsize + 1) * bsize; // number of values on x-row
	const uint bxysize = (ysize + 1) * bxsize; // number of values on xy-plane
	const uint ii0 = (IS_3D ? bxysize : 0) + bxsize + bsize; // reserved zeros in the beginning of value table
	const uint vsize = ii0 + (IS_3D ? zsize + 1 : zsize) * bxysize; // total number of values

	std::cout << "bsize: " << bsize << ", xsize: " << xsize << ", yszie: " << ysize << ", zsize: " << zsize << std::endl;
	std::cout << "bodies = " << xsize * ysize * zsize * bsize << std::endl;

	// initialize stationary state
	Buffer<Complex> Psi0(vsize, Complex(0, 0)); // initial discrete wave function
	Buffer<ddouble> pot(vsize, 0.0); // discrete potential multiplied by time step size
	ddouble g = state.getG(); // effective interaction strength
	ddouble maxpot = 0.0; // maximal value of potential
	for (k = 0; k < zsize; k++)
	{
		for (j = 0; j < ysize; j++)
		{
			for (i = 0; i < xsize; i++)
			{
				for (l = 0; l < bsize; l++)
				{
					const uint ii = ii0 + k * bxysize + j * bxsize + i * bsize + l;
					const Vector3 p(p0.x + block_scale * (i * BLOCK_WIDTH.x + bpos[l].x), p0.y + block_scale * (j * BLOCK_WIDTH.y + bpos[l].y), p0.z + block_scale * (k * BLOCK_WIDTH.z + bpos[l].z)); // position
					Psi0[ii] = state.getPsi(p);
					pot[ii] = potentialV3(p);
					const ddouble poti = pot[ii] + g * Psi0[ii].normsq();
					if (poti > maxpot) maxpot = poti;
				}
			}
		}
	}

	// Initialize device memory
	size_t dxsize = xsize + 2; // One element buffer to both ends
	size_t dysize = ysize + 2; // One element buffer to both ends
	size_t dzsize = zsize + 2; // One element buffer to both ends
	hipExtent psiExtent = make_hipExtent(dxsize * sizeof(BlockPsis), dysize, dzsize);
	hipExtent potExtent = make_hipExtent(dxsize * sizeof(BlockPots), dysize, dzsize);

	hipPitchedPtr d_cudaR;
	hipPitchedPtr d_cudaI;
	hipPitchedPtr d_cudaPot;

	checkCudaErrors(hipMalloc3D(&d_cudaR, psiExtent));
	checkCudaErrors(hipMalloc3D(&d_cudaI, psiExtent));
	checkCudaErrors(hipMalloc3D(&d_cudaPot, potExtent));

	size_t offset = d_cudaR.pitch * dysize + d_cudaR.pitch + sizeof(BlockPsis);
	size_t potOffset = d_cudaPot.pitch * dysize + d_cudaPot.pitch + sizeof(BlockPots);
	PitchedPtr d_r = { (char*)d_cudaR.ptr + offset, d_cudaR.pitch, d_cudaR.pitch * dysize };
	PitchedPtr d_i = { (char*)d_cudaI.ptr + offset, d_cudaI.pitch, d_cudaI.pitch * dysize };
	PitchedPtr d_pot = { (char*)d_cudaPot.ptr + potOffset, d_cudaPot.pitch, d_cudaPot.pitch * dysize };

	// find terms for laplacian
	Buffer<int2> lapind;
	Buffer<ddouble> hodges;
	ddouble lapfac = -0.5 * getLaplacian(lapind, hodges, sizeof(BlockPsis), d_r.pitch, d_r.slicePitch) / (block_scale * block_scale);
	const uint lapsize = lapind.size() / bsize;
	ddouble lapfac0 = lapsize * (-lapfac);

	//std::cout << "lapsize = " << lapsize << ", lapfac = " << lapfac << ", lapfac0 = " << lapfac0 << std::endl;

	// compute time step size
	const uint steps_per_iteration = 4 * uint(iteration_period * (maxpot + lapfac0)); // number of time steps per iteration period
	const ddouble time_step_size = iteration_period / ddouble(steps_per_iteration); // time step in time units

	std::cout << "steps_per_iteration = " << steps_per_iteration << std::endl;

	std::cout << "ALU operations per unit time = " << xsize * ysize * zsize * bsize * steps_per_iteration * FACE_COUNT << std::endl;

	// multiply terms with time_step_size
	g *= time_step_size;
	lapfac *= time_step_size;
	lapfac0 *= time_step_size;
	for (i = 0; i < vsize; i++) pot[i] *= time_step_size;
	for (int i = 0; i < hodges.size(); ++i) hodges[i] = -0.5 * hodges[i] / (block_scale * block_scale) * time_step_size;

	int2* d_lapind;
	checkCudaErrors(hipMalloc(&d_lapind, lapind.size() * sizeof(int2)));

	ddouble* d_hodges;
	checkCudaErrors(hipMalloc(&d_hodges, hodges.size() * sizeof(ddouble)));

	// Initialize host memory
	size_t hostSize = dxsize * dysize * (zsize + 2);
	BlockPsis* h_r;// = new BlockPsis[dxsize * dysize * (zsize + 2)];
	BlockPsis* h_i;// = new BlockPsis[dxsize * dysize * (zsize + 2)];
	BlockPots* h_pot;// = new BlockPots[dxsize * dysize * (zsize + 2)];
	checkCudaErrors(hipHostMalloc(&h_r, hostSize * sizeof(BlockPsis)));
	checkCudaErrors(hipHostMalloc(&h_i, hostSize * sizeof(BlockPsis)));
	checkCudaErrors(hipHostMalloc(&h_pot, hostSize * sizeof(BlockPots)));
	memset(h_r, 0, hostSize * sizeof(BlockPsis));
	memset(h_i, 0, hostSize * sizeof(BlockPsis));
	memset(h_pot, 0, hostSize * sizeof(BlockPots));

	// initialize discrete field
	const Complex oddPhase = state.getPhase(-1 * time_step_size);
	Random rnd(54363);
	for (k = 0; k < zsize; k++)
	{
		for (j = 0; j < ysize; j++)
		{
			for (i = 0; i < xsize; i++)
			{
				for (l = 0; l < bsize; l++)
				{
					const uint srcI = ii0 + k * bxysize + j * bxsize + i * bsize + l;
					const uint dstI = (k + 1) * dxsize * dysize + (j + 1) * dxsize + (i + 1);
					ddouble r = Psi0[srcI].r;
					ddouble i = Psi0[srcI].i;
					h_r[dstI].values[l] = r;
					h_i[dstI].values[l] = i;
					h_pot[dstI].values[l] = pot[srcI];
				}
			}
		}
	}

	hipPitchedPtr h_cudaR = { 0 };
	hipPitchedPtr h_cudaI = { 0 };
	hipPitchedPtr h_cudaPot = { 0 };

	h_cudaR.ptr = h_r;
	h_cudaR.pitch = dxsize * sizeof(BlockPsis);
	h_cudaR.xsize = d_cudaR.xsize;
	h_cudaR.ysize = d_cudaR.ysize;

	h_cudaI.ptr = h_i;
	h_cudaI.pitch = dxsize * sizeof(BlockPsis);
	h_cudaI.xsize = d_cudaI.xsize;
	h_cudaI.ysize = d_cudaI.ysize;

	h_cudaPot.ptr = h_pot;
	h_cudaPot.pitch = dxsize * sizeof(BlockPots);
	h_cudaPot.xsize = d_cudaPot.xsize;
	h_cudaPot.ysize = d_cudaPot.ysize;

	// Copy from host memory to device memory
	hipMemcpy3DParms rParams = { 0 };
	hipMemcpy3DParms iParams = { 0 };
	hipMemcpy3DParms potParams = { 0 };

	rParams.srcPtr = h_cudaR;
	rParams.dstPtr = d_cudaR;
	rParams.extent = psiExtent;
	rParams.kind = hipMemcpyHostToDevice;

	iParams.srcPtr = h_cudaI;
	iParams.dstPtr = d_cudaI;
	iParams.extent = psiExtent;
	iParams.kind = hipMemcpyHostToDevice;

	potParams.srcPtr = h_cudaPot;
	potParams.dstPtr = d_cudaPot;
	potParams.extent = potExtent;
	potParams.kind = hipMemcpyHostToDevice;

	checkCudaErrors(hipMemcpy3D(&rParams));
	checkCudaErrors(hipMemcpy3D(&iParams));
	checkCudaErrors(hipMemcpy3D(&potParams));
	checkCudaErrors(hipMemcpy(d_lapind, &lapind[0], lapind.size() * sizeof(int2), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_hodges, &hodges[0], hodges.size() * sizeof(ddouble), hipMemcpyHostToDevice));

	// Clear host memory after data has been copied to devices
	hipDeviceSynchronize();
	//Psi0.clear();
	pot.clear();
	bpos.clear();
	lapind.clear();
	hodges.clear();
	hipHostFree(h_pot);
#if !(SAVE_PICTURE || SAVE_VOLUME)
	hipHostFree(h_evenPsi);
#endif

	// Integrate in time
	uint3 dimensions = make_uint3(xsize, ysize, zsize);
	uint iter = 0;
	dim3 dimBlock(THREAD_BLOCK_X, THREAD_BLOCK_Y, THREAD_BLOCK_Z);
	dim3 dimGrid((xsize + THREAD_BLOCK_X - 1) / THREAD_BLOCK_X,
		(ysize + THREAD_BLOCK_Y - 1) / THREAD_BLOCK_Y,
		((zsize + THREAD_BLOCK_Z - 1) / THREAD_BLOCK_Z) * VALUES_IN_BLOCK);
#if SAVE_PICTURE || SAVE_VOLUME
	hipMemcpy3DParms rBackParams = { 0 };
	rBackParams.srcPtr = d_cudaR;
	rBackParams.dstPtr = h_cudaR;
	rBackParams.extent = psiExtent;
	rBackParams.kind = hipMemcpyDeviceToHost;

	hipMemcpy3DParms iBackParams = { 0 };
	iBackParams.srcPtr = d_cudaI;
	iBackParams.dstPtr = h_cudaI;
	iBackParams.extent = psiExtent;
	iBackParams.kind = hipMemcpyDeviceToHost;
#endif
	Text errorText;
	const uint time0 = clock();
	const ddouble volume = (IS_3D ? block_scale : 1.0) * block_scale * block_scale * VOLUME;
	while (true)
	{
#if SAVE_PICTURE
		// draw picture
		const float INTENSITY = 20.0f;
		const int SIZE = 2;
		int width = dxsize * SIZE, height = dysize * SIZE;
		Picture pic(width, height);
		k = zsize / 2 + 1;
		for (j = 0; j < height; j++)
		{
			for (i = 0; i < width; i++)
			{
				const uint idx = k * dxsize * dysize + (j / SIZE) * dxsize + i / SIZE;
				double norm = sqrt(h_r[idx].values[0] * h_r[idx].values[0] + h_i[idx].values[0] * h_i[idx].values[0]);
		
				pic.setColor(i, j, INTENSITY * Vector4(h_r[idx].values[0], norm, h_i[idx].values[0], 1.0));
			}
		}
		std::ostringstream picpath;
		picpath << "results/kuva" << iter << ".bmp";
		pic.save(picpath.str(), false);

		// print squared norm and error
		const Complex currentPhase = state.getPhase(iter * steps_per_iteration * time_step_size);
		ddouble errorNormSq = 0;
		ddouble normsq = 0.0;
		Complex error(0.0, 0.0);
		for (k = 0; k < zsize; k++)
		{
			for (j = 0; j < ysize; j++)
			{
				for (i = 0; i < xsize; i++)
				{
					for (l = 0; l < bsize; l++)
					{
						const uint srcI = ii0 + k * bxysize + j * bxsize + i * bsize + l;
						const uint dstI = (k + 1) * dxsize * dysize + (j + 1) * dxsize + (i + 1);

						Complex evenPsi(h_r[dstI].values[l], h_i[dstI].values[l]);
						normsq += evenPsi.normsq() * volume;
						error += (Psi0[srcI].con() * evenPsi) * volume;

						Complex groundTruth = currentPhase * Psi0[srcI];
						errorNormSq += (groundTruth - evenPsi).normsq();
					}
				}
			}
		}
		ddouble RMSE = sqrt(errorNormSq / (double)(zsize * ysize * xsize * bsize));
		ddouble errorAbs = abs(normsq - error.norm());
		std::cout << "normsq=" << normsq << " error=" << errorAbs << std::endl;
		errorText << RMSE << " ";
#endif

#if SAVE_VOLUME
		// save volume map
		const ddouble fmax = state.searchFunctionMax();
		const ddouble unit = 60000.0 / (bsize * fmax * fmax);
		Buffer<ushort> vol(dxsize * dysize * dzsize);
		for (k = 0; k < dzsize; k++)
		{
			for (j = 0; j < dysize; j++)
			{
				for (i = 0; i < dxsize; i++)
				{
					const uint idx = k * dxsize * dysize + j * dxsize + i;
					ddouble sum = 0.0;
					for (l = 0; l < bsize; l++)
					{
						sum += h_r[idx].values[0] * h_r[idx].values[0] + h_evenPsi[idx].values[0].y * h_evenPsi[idx].values[0].y;
					}
					sum *= unit;
					vol[idx] = (sum > 65535.0 ? 65535 : ushort(sum));
				}
			}
		}
		Text volpath;
		volpath << "volume" << iter << ".mhd";
		saveVolumeMap(volpath.str(), vol, dxsize, dysize, dzsize, block_scale * BLOCK_WIDTH);
#endif

		// finish iteration
		if (++iter > number_of_iterations) break;
		//++iter;
		//if (errorAbs > 0.01) break;

		// integrate one iteration
		std::cout << "Iteration " << iter << std::endl;
		for (uint step = 0; step < steps_per_iteration; step++)
		{
			// update odd values
			update << <dimGrid, dimBlock >> > (d_i, d_r, d_pot, d_lapind, d_hodges, g, dimensions, -1.0f);
			// update even values
			update << <dimGrid, dimBlock >> > (d_r, d_i, d_pot, d_lapind, d_hodges, g, dimensions, 1.0f);
		}
#if SAVE_PICTURE || SAVE_VOLUME
		// Copy back from device memory to host memory
		checkCudaErrors(hipMemcpy3D(&rBackParams));
		checkCudaErrors(hipMemcpy3D(&iBackParams));
#endif
	}
	errorText.save("results/errors.txt");

	std::cout << "iteration time = " << (1e-3 * (clock() - time0)) / number_of_iterations << std::endl;
	std::cout << "total time = " << 1e-3 * (clock() - time0) << std::endl;

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernels (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	return 0;
}

int main(int argc, char** argv)
{
#if LOAD_STATE_FROM_DISK
	VortexState state;
	state.load("state.dat");
	const ddouble eps = 1e-5 * state.searchFunctionMax();
	const ddouble maxr = state.searchMaxR(eps);
	const ddouble maxz = state.searchMaxZ(eps);
#else
	// preliminary vortex state to find vortex size
	VortexState state0;
	state0.setKappa(KAPPA);
	state0.setG(G);
	if(IS_3D) state0.setRange(0.0, 15.0, 35.0, 0.2, 0.2); // use this for 3d
	state0.iterateSolution(potentialRZ, 10000, 1e-29);
	const ddouble eps = 1e-5 * state0.searchFunctionMax();
	const ddouble minr = state0.searchMinR(eps);
	ddouble maxr = state0.searchMaxR(eps);
	ddouble maxz = state0.searchMaxZ(eps);
	//std::cout << "maxf=" << 1e6*eps << " minr=" << minr << " maxr=" << maxr << " maxz=" << maxz << std::endl;

	// more accurate vortex state
	VortexState state;
	state.setKappa(KAPPA);
	state.setG(G);
	if (IS_3D) state.setRange(minr, maxr, maxz, 0.03, 0.03); // use this for 3d
	state.initialize(state0);
	state.iterateSolution(potentialRZ, 10000, 1e-29);
	state.save("state.dat");
	maxr = state.searchMaxR(eps);
	maxz = state.searchMaxZ(eps);
	//std::cout << "maxf=" << state.searchFunctionMax() << std::endl;
#endif

	const int number_of_iterations = 100;
	const ddouble iteration_period = 1.0;
	const ddouble block_scale = PIx2 / (20.0 * sqrt(state.integrateCurvature()));

	std::cout << "1 GPU version" << std::endl;
	std::cout << "kappa = " << KAPPA << std::endl;
	std::cout << "g = " << G << std::endl;
	std::cout << "ranks = 576" << std::endl;
	std::cout << "block_scale = " << block_scale << std::endl;
	std::cout << "iteration_period = " << iteration_period << std::endl;
	std::cout << "maxr = " << maxr << std::endl;
	std::cout << "maxz = " << maxz << std::endl;
	std::cout << "dual edge length = " << DUAL_EDGE_LENGTH * block_scale << std::endl;

	// integrate in time using DEC
	if (IS_3D) integrateInTime(state, block_scale, Vector3(-maxr, -maxr, -maxz), Vector3(maxr, maxr, maxz), iteration_period, number_of_iterations); // use this for 3d
	else integrateInTime(state, block_scale, Vector3(-maxr, -maxr, 0.0), Vector3(maxr, maxr, 0.0), iteration_period, number_of_iterations); // use this for 2d

	return 0;
}
