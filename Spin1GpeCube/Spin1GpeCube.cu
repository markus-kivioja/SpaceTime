#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "VortexState.hpp"
#include <Output/Picture.hpp>
#include <Output/Text.hpp>
#include <Types/Complex.hpp>
#include <Types/Random.hpp>
#include <Mesh/DelaunayMesh.hpp>
#include <iostream>
#include <sstream>

#include <mesh.h>

constexpr double Lx = 24.0;
constexpr double Ly = 24.0;
constexpr double Lz = 24.0;

constexpr double Nx = 200.0;
constexpr double Ny = 200.0;
constexpr double Nz = 200.0;

constexpr double omega_r = 160 * 2 * PI;
constexpr double omega_z = 220 * 2 * PI;
constexpr double lambda_x = 1.0;
constexpr double lambda_y = 1.0;
constexpr double lambda_z = omega_z / omega_r;

constexpr double c0 = 14161.2119140625;
constexpr double c2 = -65.5179061889648;

constexpr double hbar = 1.05457148e-34; // [m^2 kg / s]
const double a_r = sqrt(hbar / ((1.44316060e-25) * omega_r)); //[m]
constexpr double muB = 9.27400968e-24; // [m^2 kg / s^2 T^-1] bohr magneton

const double BqScale = -(0.5 * muB / (hbar * omega_r) * a_r) / 100.; // [cm/Gauss]
constexpr double Bz0Scale = -(0.5 * muB / (hbar * omega_r)) / 10000.; // [1/Gauss]

constexpr double dimensionalBq = 3.7;
constexpr double dimensionalBz0 = 0.01;

// The external magnetic field
const double Bq = dimensionalBq * BqScale;
const double Bz0 = dimensionalBz0 * Bz0Scale;
constexpr double BzVel = -1.0 * Bz0Scale / omega_r;
constexpr double  Bzf = -0.0006;

const std::string STATE_FILENAME = "ground_state_double.dat";

#define INV_SQRT_2 0.70710678118655

#define COMPUTE_GROUND_STATE 1
#define FORCE_SPIN_POLARISATION 0

#define SAVE_PICTURE 0
#define SAVE_VOLUME 0
#define SAVE_FREQUENCY 1000

#define THREAD_BLOCK_X 8
#define THREAD_BLOCK_Y 8
#define THREAD_BLOCK_Z 1

__host__ __device__ __inline__ double trap(double3 p)
{
	double x = p.x * lambda_x;
	double y = p.y * lambda_y;
	double z = p.z * lambda_z;
	return 0.5 * (x * x + y * y + z * z) + 100.0;
}

__host__ __device__ __inline__ double3 magneticField(double3 p, double Bq, double Bz)
{
	return make_double3(Bq * p.x, Bq * p.y, Bq * -2 * p.z + Bz);
}

#include <utils.h>

__global__ void density(double* density, PitchedPtr prevStep, uint3 dimensions, double dv)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataZid = zid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (xid >= dimensions.x || yid >= dimensions.y || dataZid >= dimensions.z)
	{
		return;
	}

	size_t dualNodeId = zid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	char* pPsi = prevStep.ptr + prevStep.slicePitch * dataZid + prevStep.pitch * yid + sizeof(BlockPsis) * xid;
	Complex3Vec psi = ((BlockPsis*)pPsi)->values[dualNodeId];

	size_t idx = dataZid * dimensions.x * dimensions.y * VALUES_IN_BLOCK + yid * dimensions.x * VALUES_IN_BLOCK + xid * VALUES_IN_BLOCK + dualNodeId;
	density[idx] = dv * ((psi.s1 * star(psi.s1)).x + (psi.s0 * star(psi.s0)).x + (psi.s_1 * star(psi.s_1)).x);
}

__global__ void integrate(double* dataVec, size_t stride, bool addLast)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= stride)
	{
		return;
	}

	dataVec[idx] += dataVec[idx + stride];

	if ((idx == (stride - 1)) && addLast)
	{
		dataVec[idx] += dataVec[idx + stride + 1];
	}
}

__global__ void normalize(double* density, PitchedPtr psiPtr, uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataZid = zid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (xid >= dimensions.x || yid >= dimensions.y || dataZid >= dimensions.z)
	{
		return;
	}

	size_t dualNodeId = zid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)
	BlockPsis* blockPsis = (BlockPsis*)(psiPtr.ptr + psiPtr.slicePitch * dataZid + psiPtr.pitch * yid) + xid;
	Complex3Vec psi = blockPsis->values[dualNodeId];
	double sqrtDens = sqrt(density[0]);
	psi.s1 = psi.s1 / sqrtDens;
	psi.s0 = psi.s0 / sqrtDens;
	psi.s_1 = psi.s_1 / sqrtDens;

	blockPsis->values[dualNodeId] = psi;
}

#if COMPUTE_GROUND_STATE
__global__ void itp(PitchedPtr nextStep, PitchedPtr prevStep, int2* __restrict__ lapInd, double* __restrict__ hodges, double Bq, double Bz, uint3 dimensions, double block_scale, double3 p0, double dt)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;

	size_t dataZid = zid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (xid >= dimensions.x || yid >= dimensions.y || dataZid >= dimensions.z)
	{
		return;
	}

	// Calculate the pointers for this block
	char* prevPsi = prevStep.ptr + prevStep.slicePitch * dataZid + prevStep.pitch * yid + sizeof(BlockPsis) * xid;
	BlockPsis* nextPsi = (BlockPsis*)(nextStep.ptr + nextStep.slicePitch * dataZid + nextStep.pitch * yid) + xid;

	// Update psi
	size_t dualNodeId = zid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)
	Complex3Vec prev = ((BlockPsis*)prevPsi)->values[dualNodeId];

	uint primaryFace = dualNodeId * FACE_COUNT;

	Complex3Vec H;
	H.s1 = make_double2(0, 0);
	H.s0 = make_double2(0, 0);
	H.s_1 = make_double2(0, 0);

#pragma unroll
	for (int i = 0; i < FACE_COUNT; ++i)
	{
		Complex3Vec otherBoundaryZeroCell = ((BlockPsis*)(prevPsi + lapInd[primaryFace].x))->values[lapInd[primaryFace].y];
		H.s1 += hodges[primaryFace] * (otherBoundaryZeroCell.s1 - prev.s1);
		H.s0 += hodges[primaryFace] * (otherBoundaryZeroCell.s0 - prev.s0);
		H.s_1 += hodges[primaryFace] * (otherBoundaryZeroCell.s_1 - prev.s_1);

		primaryFace++;
	}

	double normSq_s1 = prev.s1.x * prev.s1.x + prev.s1.y * prev.s1.y;
	double normSq_s0 = prev.s0.x * prev.s0.x + prev.s0.y * prev.s0.y;
	double normSq_s_1 = prev.s_1.x * prev.s_1.x + prev.s_1.y * prev.s_1.y;
	double normSq = normSq_s1 + normSq_s0 + normSq_s_1;
	
	// Add the total potential to Hamiltonian
	double3 localPos = getLocalPos(dualNodeId);
	double3 globalPos = make_double3(p0.x + block_scale * (xid * BLOCK_WIDTH_X + localPos.x),
		p0.y + block_scale * (yid * BLOCK_WIDTH_Y + localPos.y),
		p0.z + block_scale * (dataZid * BLOCK_WIDTH_Z + localPos.z));
	double totalPot = trap(globalPos) + (c0 + c2) * normSq;

	H.s1 += totalPot * prev.s1 + c2 * (-2.0 * normSq_s_1 * prev.s1 + star(prev.s_1) * prev.s0 * prev.s0 + 0 * prev.s_1);
	H.s0 += totalPot * prev.s0 + c2 * (star(prev.s0) * prev.s_1 * prev.s1 - normSq_s0 * prev.s0 + star(prev.s0) * prev.s1 * prev.s_1);
	H.s_1 += totalPot * prev.s_1 + c2 * (0 * prev.s1 + star(prev.s1) * prev.s0 * prev.s0 - 2.0 * normSq_s1 * prev.s_1);

	// Add the Zeeman term
	double3 B = magneticField(globalPos, Bq, Bz);
	double2 Bxy = INV_SQRT_2 * make_double2(B.x, B.y);
	double2 Bxy_star = star(Bxy);
	
	H.s1 += (B.z * prev.s1 + Bxy_star * prev.s0);
	H.s0 += (Bxy * prev.s1 + Bxy_star * prev.s_1);
	H.s_1 += (Bxy * prev.s0 - B.z * prev.s_1);

#if FORCE_SPIN_POLARISATION
	nextPsi->values[dualNodeId].s1 = prev.s1 - dt * make_double2(H.s1.x, H.s1.y);
	nextPsi->values[dualNodeId].s0 = make_double2(0.0, 0.0);
	nextPsi->values[dualNodeId].s_1 = make_double2(0.0, 0.0);
#else
	nextPsi->values[dualNodeId].s1 = prev.s1 - dt * make_double2(H.s1.x, H.s1.y);
	nextPsi->values[dualNodeId].s0 = prev.s0 - dt * make_double2(H.s0.x, H.s0.y);
	nextPsi->values[dualNodeId].s_1 =prev.s_1 - dt * make_double2(H.s_1.x, H.s_1.y);
#endif
};
#else
__global__ void update(PitchedPtr nextStep, PitchedPtr prevStep, int2* __restrict__ lapInd, double* __restrict__ hodges, double Bq, double Bz, uint3 dimensions, double block_scale, double3 p0, double dt)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;

	size_t dataZid = zid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (xid >= dimensions.x || yid >= dimensions.y || dataZid >= dimensions.z)
	{
		return;
	}

	// Calculate the pointers for this block
	char* prevPsi = prevStep.ptr + prevStep.slicePitch * dataZid + prevStep.pitch * yid + sizeof(BlockPsis) * xid;
	BlockPsis* nextPsi = (BlockPsis*)(nextStep.ptr + nextStep.slicePitch * dataZid + nextStep.pitch * yid) + xid;

	// Update psi
	size_t dualNodeId = zid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)
	Complex3Vec prev = ((BlockPsis*)prevPsi)->values[dualNodeId];

	uint primaryFace = dualNodeId * FACE_COUNT;

	Complex3Vec H;
	H.s1 = make_double2(0, 0);
	H.s0 = make_double2(0, 0);
	H.s_1 = make_double2(0, 0);

#pragma unroll
	for (int i = 0; i < FACE_COUNT; ++i)
	{
		Complex3Vec otherBoundaryZeroCell = ((BlockPsis*)(prevPsi + lapInd[primaryFace].x))->values[lapInd[primaryFace].y];
		H.s1 += hodges[primaryFace] * (otherBoundaryZeroCell.s1 - prev.s1);
		H.s0 += hodges[primaryFace] * (otherBoundaryZeroCell.s0 - prev.s0);
		H.s_1 += hodges[primaryFace] * (otherBoundaryZeroCell.s_1 - prev.s_1);

		primaryFace++;
	}

	double normSq_s1 = prev.s1.x * prev.s1.x + prev.s1.y * prev.s1.y;
	double normSq_s0 = prev.s0.x * prev.s0.x + prev.s0.y * prev.s0.y;
	double normSq_s_1 = prev.s_1.x * prev.s_1.x + prev.s_1.y * prev.s_1.y;
	double normSq = normSq_s1 + normSq_s0 + normSq_s_1;

	// Add the total potential to Hamiltonian
	double3 localPos = getLocalPos(dualNodeId);
	double3 globalPos = make_double3(p0.x + block_scale * (xid * BLOCK_WIDTH_X + localPos.x),
		p0.y + block_scale * (yid * BLOCK_WIDTH_Y + localPos.y),
		p0.z + block_scale * (dataZid * BLOCK_WIDTH_Z + localPos.z));
	double totalPot = trap(globalPos) + (c0 + c2) * normSq;

	H.s1 += totalPot * prev.s1 + c2 * (-2.0 * normSq_s_1 * prev.s1 + star(prev.s_1) * prev.s0 * prev.s0 + 0 * prev.s_1);
	H.s0 += totalPot * prev.s0 + c2 * (star(prev.s0) * prev.s_1 * prev.s1 - normSq_s0 * prev.s0 + star(prev.s0) * prev.s1 * prev.s_1);
	H.s_1 += totalPot * prev.s_1 + c2 * (0 * prev.s1 + star(prev.s1) * prev.s0 * prev.s0 - 2.0 * normSq_s1 * prev.s_1);

	// Add the Zeeman term
	double3 B = magneticField(globalPos, Bq, Bz);
	double2 Bxy = INV_SQRT_2 * make_double2(B.x, B.y);
	double2 Bxy_star = star(Bxy);

	H.s1 += (B.z * prev.s1 + Bxy_star * prev.s0);
	H.s0 += (Bxy * prev.s1 + Bxy_star * prev.s_1);
	H.s_1 += (Bxy * prev.s0 - B.z * prev.s_1);

	nextPsi->values[dualNodeId].s1 += dt * make_double2(H.s1.y, -H.s1.x);
	nextPsi->values[dualNodeId].s0 += dt * make_double2(H.s0.y, -H.s0.x);
	nextPsi->values[dualNodeId].s_1 += dt * make_double2(H.s_1.y, -H.s_1.x);
};
#endif

//void energy_h(dim3 dimGrid, dim3 dimBlock, double* energyPtr, PitchedPtr psi, PitchedPtr potentials, int2* lapInd, double* hodges, double g, uint3 dimensions, double volume, size_t bodies)
//{
//	energy << <dimGrid, dimBlock >> > (energyPtr, psi, potentials, lapInd, hodges, g, dimensions, volume);
//	int prevStride = bodies;
//	while (prevStride > 1)
//	{
//		int newStride = prevStride / 2;
//		integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (energyPtr, newStride, ((newStride * 2) != prevStride));
//		prevStride = newStride;
//	}
//}

void normalize_h(dim3 dimGrid, dim3 dimBlock, double* densityPtr, PitchedPtr psi, uint3 dimensions, size_t bodies, double volume)
{
	density << <dimGrid, dimBlock >> > (densityPtr, psi, dimensions, volume);
	int prevStride = bodies;
	while (prevStride > 1)
	{
		int newStride = prevStride / 2;
		integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (densityPtr, newStride, ((newStride * 2) != prevStride));
		prevStride = newStride;
	}

	normalize << < dimGrid, dimBlock >> > (densityPtr, psi, dimensions);
}

void printDensity(dim3 dimGrid, dim3 dimBlock, double* densityPtr, PitchedPtr psi, uint3 dimensions, size_t bodies, double volume)
{
	density << <dimGrid, dimBlock >> > (densityPtr, psi, dimensions, volume);
	int prevStride = bodies;
	while (prevStride > 1)
	{
		int newStride = prevStride / 2;
		integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (densityPtr, newStride, ((newStride * 2) != prevStride));
		prevStride = newStride;
	}
	double hDensity = 0;
	checkCudaErrors(hipMemcpy(&hDensity, densityPtr, sizeof(double), hipMemcpyDeviceToHost));

	std::cout << "Total density: " << hDensity << std::endl;
}

uint integrateInTime(const double block_scale, const Vector3& minp, const Vector3& maxp, const double iteration_period, const uint number_of_iterations)
{
	uint i, j, k, l;

	// find dimensions
	const Vector3 domain = maxp - minp;
	const uint xsize = uint(domain.x / (block_scale * BLOCK_WIDTH.x)) + 1;
	const uint ysize = uint(domain.y / (block_scale * BLOCK_WIDTH.y)) + 1;
	const uint zsize = uint(domain.z / (block_scale * BLOCK_WIDTH.z)) + 1;
	const Vector3 p0 = 0.5 * (minp + maxp - block_scale * Vector3(BLOCK_WIDTH.x * xsize, BLOCK_WIDTH.y * ysize, BLOCK_WIDTH.z * zsize));
	std::cout << p0.x << ", " << p0.y << ", " << p0.z << std::endl;
	const double3 d_p0 = make_double3(p0.x, p0.y, p0.z);

	std::cout << xsize << ", " << ysize << ", " << zsize << std::endl;

	// compute discrete dimensions
	const uint bsize = VALUES_IN_BLOCK; // bpos.size(); // number of values inside a block
	const uint bxsize = (xsize + 1) * bsize; // number of values on x-row
	const uint bxysize = (ysize + 1) * bxsize; // number of values on xy-plane
	const uint ii0 = bxysize + bxsize + bsize; // reserved zeros in the beginning of value table
	const uint vsize = ii0 + (zsize + 1) * bxysize; // total number of values

	std::cout << "bsize: " << bsize << ", xsize: " << xsize << ", yszie: " << ysize << ", zsize: " << zsize << std::endl;
	uint64_t bodies = xsize * ysize * zsize * bsize;
	std::cout << "bodies = " << bodies << std::endl;

	// Initialize device memory
	size_t dxsize = xsize + 2; // One element buffer to both ends
	size_t dysize = ysize + 2; // One element buffer to both ends
	size_t dzsize = zsize + 2; // One element buffer to both ends
	hipExtent psiExtent = make_hipExtent(dxsize * sizeof(BlockPsis), dysize, dzsize);

	hipPitchedPtr d_cudaEvenPsi;
	hipPitchedPtr d_cudaOddPsi;
	checkCudaErrors(hipMalloc3D(&d_cudaEvenPsi, psiExtent));
	checkCudaErrors(hipMalloc3D(&d_cudaOddPsi, psiExtent));

	//double* d_energy;
	double* d_density;
	//checkCudaErrors(hipMalloc(&d_energy, bodies * sizeof(double)));
	checkCudaErrors(hipMalloc(&d_density, bodies * sizeof(double)));

	size_t offset = d_cudaEvenPsi.pitch * dysize + d_cudaEvenPsi.pitch + sizeof(BlockPsis);
	PitchedPtr d_evenPsi = { (char*)d_cudaEvenPsi.ptr + offset, d_cudaEvenPsi.pitch, d_cudaEvenPsi.pitch * dysize };
	PitchedPtr d_oddPsi = { (char*)d_cudaOddPsi.ptr + offset, d_cudaOddPsi.pitch, d_cudaOddPsi.pitch * dysize };

	// find terms for laplacian
	Buffer<int2> lapind;
	Buffer<double> hodges;
	double lapfac = -0.5 * getLaplacian(lapind, hodges, sizeof(BlockPsis), d_evenPsi.pitch, d_evenPsi.slicePitch) / (block_scale * block_scale);
	const uint lapsize = lapind.size() / bsize;
	double lapfac0 = lapsize * (-lapfac);

	//std::cout << "lapsize = " << lapsize << ", lapfac = " << lapfac << ", lapfac0 = " << lapfac0 << std::endl;

	// compute time step size
	const uint steps_per_iteration = 100; // 1.0 / 0.000199999994947575; // uint(iteration_period * (maxpot + lapfac0)) + 1; // number of time steps per iteration period
	const double dt = 0.000199999994947575; // iteration_period / double(steps_per_iteration); // time step in time units

	std::cout << "steps_per_iteration = " << steps_per_iteration << std::endl;

	std::cout << "ALU operations per unit time = " << xsize * ysize * zsize * bsize * steps_per_iteration * FACE_COUNT << std::endl;

	for (int i = 0; i < hodges.size(); ++i) hodges[i] = -0.5 * hodges[i] / (block_scale * block_scale);

	int2* d_lapind;
	checkCudaErrors(hipMalloc(&d_lapind, lapind.size() * sizeof(int2)));

	double* d_hodges;
	checkCudaErrors(hipMalloc(&d_hodges, hodges.size() * sizeof(double)));

	// Initialize host memory
	size_t hostSize = dxsize * dysize * (zsize + 2);
	BlockPsis* h_evenPsi;// = new BlockPsis[dxsize * dysize * (zsize + 2)];
	BlockPsis* h_oddPsi;// = new BlockPsis[dxsize * dysize * (zsize + 2)];
	checkCudaErrors(hipHostMalloc(&h_evenPsi, hostSize * sizeof(BlockPsis)));
	checkCudaErrors(hipHostMalloc(&h_oddPsi, hostSize * sizeof(BlockPsis)));
	memset(h_evenPsi, 0, hostSize * sizeof(BlockPsis));
	memset(h_oddPsi, 0, hostSize * sizeof(BlockPsis));

#if COMPUTE_GROUND_STATE
	// Initialize discrete field
	Random rnd(54363);
	for (k = 0; k < zsize; k++)
	{
		for (j = 0; j < ysize; j++)
		{
			for (i = 0; i < xsize; i++)
			{
				for (l = 0; l < bsize; l++)
				{
					const uint srcI = ii0 + k * bxysize + j * bxsize + i * bsize + l;
					const uint dstI = (k + 1) * dxsize * dysize + (j + 1) * dxsize + (i + 1);
					const Vector2 s1 = rnd.getUniformCircle();
					const Vector2 s0 = rnd.getUniformCircle();
					const Vector2 s_1 = rnd.getUniformCircle();
					h_evenPsi[dstI].values[l].s1 = make_double2(s1.x, s1.y);
					h_evenPsi[dstI].values[l].s0 = make_double2(s0.x, s0.y);
					h_evenPsi[dstI].values[l].s_1 = make_double2(s_1.x, s_1.y);
				}
			}
		}
	}
#else
	std::ifstream fs(STATE_FILENAME, std::ios::binary | std::ios::in);
	if (fs.fail() != 0)
	{
		std::cout << "Failed to open file " << STATE_FILENAME << std::endl;
		return 1;
	}
	fs.read((char*)&h_evenPsi[0], hostSize * sizeof(BlockPsis));
	memcpy(&h_oddPsi[0], &h_evenPsi[0], hostSize * sizeof(BlockPsis));
	fs.close();
#endif

	hipPitchedPtr h_cudaEvenPsi = { 0 };
	hipPitchedPtr h_cudaOddPsi = { 0 };

	h_cudaEvenPsi.ptr = h_evenPsi;
	h_cudaEvenPsi.pitch = dxsize * sizeof(BlockPsis);
	h_cudaEvenPsi.xsize = d_cudaEvenPsi.xsize;
	h_cudaEvenPsi.ysize = d_cudaEvenPsi.ysize;

	h_cudaOddPsi.ptr = h_oddPsi;
	h_cudaOddPsi.pitch = dxsize * sizeof(BlockPsis);
	h_cudaOddPsi.xsize = d_cudaOddPsi.xsize;
	h_cudaOddPsi.ysize = d_cudaOddPsi.ysize;

	// Copy from host memory to device memory
	hipMemcpy3DParms evenPsiParams = { 0 };
	hipMemcpy3DParms oddPsiParams = { 0 };

	evenPsiParams.srcPtr = h_cudaEvenPsi;
	evenPsiParams.dstPtr = d_cudaEvenPsi;
	evenPsiParams.extent = psiExtent;
	evenPsiParams.kind = hipMemcpyHostToDevice;

	oddPsiParams.srcPtr = h_cudaOddPsi;
	oddPsiParams.dstPtr = d_cudaOddPsi;
	oddPsiParams.extent = psiExtent;
	oddPsiParams.kind = hipMemcpyHostToDevice;

	checkCudaErrors(hipMemcpy3D(&evenPsiParams));
	checkCudaErrors(hipMemcpy3D(&oddPsiParams));
	checkCudaErrors(hipMemcpy(d_lapind, &lapind[0], lapind.size() * sizeof(int2), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_hodges, &hodges[0], hodges.size() * sizeof(double), hipMemcpyHostToDevice));

	// Clear host memory after data has been copied to devices
	hipDeviceSynchronize();
	lapind.clear();
	hodges.clear();
	hipHostFree(h_oddPsi);
#if !(SAVE_PICTURE || SAVE_VOLUME)
	hipHostFree(h_evenPsi);
#endif

	// Integrate in time
	uint3 dimensions = make_uint3(xsize, ysize, zsize);
	uint iter = 0;
	dim3 dimBlock(THREAD_BLOCK_X, THREAD_BLOCK_Y, THREAD_BLOCK_Z);
	dim3 dimGrid((xsize + THREAD_BLOCK_X - 1) / THREAD_BLOCK_X,
		(ysize + THREAD_BLOCK_Y - 1) / THREAD_BLOCK_Y,
		((zsize + THREAD_BLOCK_Z - 1) / THREAD_BLOCK_Z) * VALUES_IN_BLOCK);

	hipMemcpy3DParms evenPsiBackParams = { 0 };
	evenPsiBackParams.srcPtr = d_cudaEvenPsi;
	evenPsiBackParams.dstPtr = h_cudaEvenPsi;
	evenPsiBackParams.extent = psiExtent;
	evenPsiBackParams.kind = hipMemcpyDeviceToHost;

	const double volume = block_scale * block_scale * block_scale * VOLUME;
	double t = 0;

#if COMPUTE_GROUND_STATE
	normalize_h(dimGrid, dimBlock, d_density, d_evenPsi, dimensions, bodies, volume);

	double mu = 0;
	double E = 1e20;
	auto Hpsi = d_oddPsi;
	while (true)
	{
		if ((iter % 1000) == 0) std::cout << "Iteration " << iter << std::endl;
		if (iter == 50000)
		{
			checkCudaErrors(hipMemcpy3D(&evenPsiBackParams));
			std::ofstream fs(STATE_FILENAME, std::ios::binary | std::ios_base::trunc);
			if (fs.fail() != 0) return 1;
			fs.write((char*)&h_evenPsi[0], hostSize * sizeof(BlockPsis));
			fs.close();
			return 0;
		}
#if SAVE_PICTURE
		if ((iter % SAVE_FREQUENCY) == 0)
		{
			checkCudaErrors(hipMemcpy3D(&evenPsiBackParams));
			double Bz = Bz0 + BzVel * t;
			drawPicture("GS", h_evenPsi, dxsize, dysize, dzsize, iter, Bq, Bz, block_scale, d_p0);
			printDensity(dimGrid, dimBlock, d_density, d_evenPsi, dimensions, bodies, volume);
		}
#endif
		double Bz = Bz0 + BzVel * t;
		// Take an imaginary time step
		itp << <dimGrid, dimBlock >> > (d_oddPsi, d_evenPsi, d_lapind, d_hodges, Bq, Bz, dimensions, block_scale, d_p0, dt);
		// Normalize
		normalize_h(dimGrid, dimBlock, d_density, d_oddPsi, dimensions, bodies, volume);
		t += dt;

		Bz = Bz0 + BzVel * t;
		// Take an imaginary time step
		itp << <dimGrid, dimBlock >> > (d_evenPsi, d_oddPsi, d_lapind, d_hodges, Bq, Bz, dimensions, block_scale, d_p0, dt);
		// Normalize
		normalize_h(dimGrid, dimBlock, d_density, d_evenPsi, dimensions, bodies, volume);
		t += dt;

		//energy_h(dimGrid, dimBlock, d_energy, d_evenPsi, d_pot, d_lapind, d_hodges, g, dimensions, volume, bodies);
		//double hDensity = 0;
		//double hEnergy = 0;
		//checkCudaErrors(hipMemcpy(&hDensity, d_density, sizeof(double), hipMemcpyDeviceToHost));
		//checkCudaErrors(hipMemcpy(&hEnergy, d_energy, sizeof(double), hipMemcpyDeviceToHost));

		//double newMu = hEnergy / hDensity;
		//double newE = hEnergy;
		//
		//std::cout << "Total density: " << hDensity << ", Total energy: " << hEnergy << ", mu: " << newMu << std::endl;

		//if (std::abs(mu - newMu) < 1e-4) break;

		//mu = newMu;
		//E = newE;

		iter++;
	}

#else
	Text errorText;
	const uint time0 = clock();
	while (true)
	{
#if SAVE_PICTURE
		{
			double Bz = Bz0 + BzVel * t;
			Bz = min(Bz, -0.001 * Bz0Scale);
			drawPicture("TI", h_evenPsi, dxsize, dysize, dzsize, iter, Bq, Bz, block_scale, d_p0);
			printDensity(dimGrid, dimBlock, d_density, d_evenPsi, dimensions, bodies, volume);
		}
#endif
#if SAVE_VOLUME
		saveVolue(h_evenPsi, dxsize, dysize, dzsize, iter);
#endif

		// finish iteration
		if (++iter > number_of_iterations) break;
		//++iter;
		//if (errorAbs > 0.01) break;

		// integrate one iteration
		std::cout << "Iteration " << iter << std::endl;
		for (uint step = 0; step < steps_per_iteration; step++)
		{
			double Bz = Bz0 + BzVel * t;
			Bz = min(Bz, Bzf * Bz0Scale);

			// update odd values
			update << <dimGrid, dimBlock >> > (d_oddPsi, d_evenPsi, d_lapind, d_hodges, Bq, Bz, dimensions, block_scale, d_p0, dt);
			t += dt;

			Bz = Bz0 + BzVel * t;
			Bz = min(Bz, Bzf * Bz0Scale);

			// update even values
			update << <dimGrid, dimBlock >> > (d_evenPsi, d_oddPsi, d_lapind, d_hodges, Bq, Bz, dimensions, block_scale, d_p0, dt);
			t += dt;
		}

#if SAVE_PICTURE || SAVE_VOLUME
		// Copy back from device memory to host memory
		checkCudaErrors(hipMemcpy3D(&evenPsiBackParams));
#endif
	}
	errorText.save("results/errors.txt");

	std::cout << "iteration time = " << (1e-3 * (clock() - time0)) / number_of_iterations << std::endl;
	std::cout << "total time = " << 1e-3 * (clock() - time0) << std::endl;
#endif

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernels (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	return 0;
}

int main(int argc, char** argv)
{
	const int number_of_iterations = 1000000;
	const double iteration_period = 1.0;
	const double block_scale = Lx / Nx;
	
	std::cout << "block_scale = " << block_scale << std::endl;
	std::cout << "iteration_period = " << iteration_period << std::endl;
	std::cout << "dual edge length = " << DUAL_EDGE_LENGTH * block_scale << std::endl;
	
	// integrate in time using DEC
	auto domainMin = Vector3(-Lx * 0.5, -Ly * 0.5, -Lz * 0.5);
	auto domainMax = Vector3(Lx * 0.5, Ly * 0.5, Lz * 0.5);
	integrateInTime(block_scale, domainMin, domainMax, iteration_period, number_of_iterations);

	return 0;
}
