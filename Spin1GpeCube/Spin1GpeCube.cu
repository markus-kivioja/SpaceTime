#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include "VortexState.hpp"
#include <Output/Picture.hpp>
#include <Output/Text.hpp>
#include <Types/Complex.hpp>
#include <Types/Random.hpp>
#include <Mesh/DelaunayMesh.hpp>
#include <iostream>
#include <sstream>

#include <mesh.h>

constexpr ddouble Lx = 24.0;
constexpr ddouble Ly = 24.0;
constexpr ddouble Lz = 24.0;

constexpr ddouble Nx = 200.0;
constexpr ddouble Ny = 200.0;
constexpr ddouble Nz = 200.0;

constexpr ddouble omega_r = 160 * 2 * PI;
constexpr ddouble omega_z = 220 * 2 * PI;
__constant__ ddouble lambda_x = 1.0;
__constant__ ddouble lambda_y = 1.0;
__constant__ ddouble lambda_z = omega_z / omega_r;

__constant__ ddouble c0 = 14161.2119140625;
__constant__ ddouble c2 = -65.5179061889648;

// The external magnetic field
__constant__ ddouble Bq = -1.37972986698151;
__constant__ ddouble Bz0 = -43.7382698059082;
__constant__ ddouble BzVelocity = 0; // Time derivative of the bias field TODO: Set

#define INV_SQRT_2 0.70710678118655

#define COMPUTE_GROUND_STATE 1

#define SAVE_PICTURE 1
#define SAVE_VOLUME 0
#define SAVE_FREQUENCY 100

#define THREAD_BLOCK_X 8
#define THREAD_BLOCK_Y 8
#define THREAD_BLOCK_Z 1

__host__ __device__ __inline__ ddouble trap(double3 p)
{
	ddouble x = p.x * lambda_x;
	ddouble y = p.y * lambda_y;
	ddouble z = p.z * lambda_z;
	return 0.5 * (x * x + y * y + z * z) + 100.0;
}

__device__ __inline__ double3 magneticField(double3 p, ddouble t)
{
	ddouble Bz = Bz0 + BzVelocity * t;

	return make_double3(Bq * p.x, Bq * p.y, Bq * -2 * p.z + Bz);
}

bool saveVolumeMap(const std::string& path, const Buffer<ushort>& vol, const uint xsize, const uint ysize, const uint zsize, const Vector3& h)
{
	Text rawpath;
	rawpath << path << ".raw";

	// save raw
	std::ofstream fs(rawpath.str().c_str(), std::ios_base::binary | std::ios::trunc);
	if (fs.fail()) return false;
	fs.write((char*)&vol[0], 2 * xsize * ysize * zsize);
	fs.close();

	// save header
	Text text;

	text << "ObjectType              = Image" << std::endl;
	text << "NDims                   = 3" << std::endl;
	text << "BinaryData              = True" << std::endl;
	text << "CompressedData          = False" << std::endl;
	text << "BinaryDataByteOrderMSB  = False" << std::endl;
	text << "TransformMatrix         = 1 0 0 0 1 0 0 0 1" << std::endl;
	text << "Offset                  = " << -0.5 * xsize * h.x << " " << -0.5 * ysize * h.y << " " << -0.5 * zsize * h.z << std::endl;
	text << "CenterOfRotation        = 0 0 0" << std::endl;
	text << "DimSize                 = " << xsize << " " << ysize << " " << zsize << std::endl;
	text << "ElementSpacing          = " << h.x << " " << h.y << " " << h.z << std::endl;
	text << "ElementNumberOfChannels = 1" << std::endl;
	text << "ElementType             = MET_USHORT" << std::endl;
	text << "ElementDataFile         = " << rawpath.str() << std::endl;
	text.save(path);
	return true;
}

struct Complex3Vec
{
	double2 s1 = make_double2(0, 0);
	double2 s0 = make_double2(0, 0);
	double2 s_1 = make_double2(0, 0);
};

struct BlockPsis
{
	Complex3Vec values[VALUES_IN_BLOCK];
};

struct PitchedPtr
{
	char* __restrict__ ptr;
	size_t pitch;
	size_t slicePitch;
};

// Arithmetic operators for cuda vector types
inline __host__ __device__ __inline__ double2 operator+(double2 a, double2 b)
{
	return make_double2(a.x + b.x, a.y + b.y);
}
inline __host__ __device__ __inline__ double2 operator-(double2 a, double2 b)
{
	return make_double2(a.x - b.x, a.y - b.y);
}
inline __host__ __device__ __inline__ void operator+=(double2& a, double2 b)
{
	a.x += b.x;
	a.y += b.y;
}
inline __host__ __device__ __inline__ void operator-=(double2& a, double2 b)
{
	a.x -= b.x;
	a.y -= b.y;
}
inline __host__ __device__ __inline__ double2 operator*(double b, double2 a)
{
	return make_double2(b * a.x, b * a.y);
}
inline __host__ __device__ __inline__ double2 operator/(double2 a, double b)
{
	return make_double2(a.x / b, a.y / b);
}
inline __host__ __device__ __inline__ double2 star(double2 a) // Complex conjugate
{
	return make_double2(a.x, -a.y);
}
inline __host__ __device__ __inline__ double2 operator*(double2 a, double2 b) // Complex number multiplication
{
	return make_double2(a.x * b.x - a.y * b.y, a.y * b.x + a.x * b.y);
}

__global__ void density(ddouble* density, PitchedPtr prevStep, uint3 dimensions, ddouble dv)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataZid = zid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (xid >= dimensions.x || yid >= dimensions.y || dataZid >= dimensions.z)
	{
		return;
	}

	size_t dualNodeId = zid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	char* pPsi = prevStep.ptr + prevStep.slicePitch * dataZid + prevStep.pitch * yid + sizeof(BlockPsis) * xid;
	Complex3Vec psi = ((BlockPsis*)pPsi)->values[dualNodeId];

	size_t idx = dataZid * dimensions.x * dimensions.y * VALUES_IN_BLOCK + yid * dimensions.x * VALUES_IN_BLOCK + xid * VALUES_IN_BLOCK + dualNodeId;
	density[idx] = dv * ((psi.s1 * star(psi.s1)).x + (psi.s0 * star(psi.s0)).x + (psi.s_1 * star(psi.s_1)).x);
}

__global__ void integrate(ddouble* dataVec, size_t stride, bool addLast)
{
	size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

	if (idx >= stride)
	{
		return;
	}

	dataVec[idx] += dataVec[idx + stride];

	if ((idx == (stride - 1)) && addLast)
	{
		dataVec[idx] += dataVec[idx + stride + 1];
	}
}

__global__ void normalize(ddouble* density, PitchedPtr psiPtr, uint3 dimensions)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;
	size_t dataZid = zid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (xid >= dimensions.x || yid >= dimensions.y || dataZid >= dimensions.z)
	{
		return;
	}

	size_t dualNodeId = zid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)
	BlockPsis* blockPsis = (BlockPsis*)(psiPtr.ptr + psiPtr.slicePitch * dataZid + psiPtr.pitch * yid) + xid;
	Complex3Vec psi = blockPsis->values[dualNodeId];
	double sqrtDens = sqrt(density[0]);
	psi.s1 = psi.s1 / sqrtDens;
	psi.s0 = psi.s0 / sqrtDens;
	psi.s_1 = psi.s_1 / sqrtDens;

	blockPsis->values[dualNodeId] = psi;
}

#if COMPUTE_GROUND_STATE
__global__ void itp(PitchedPtr nextStep, PitchedPtr prevStep, int2* __restrict__ lapInd, double* __restrict__ hodges, uint3 dimensions, double block_scale, double3 p0, double dt)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;

	size_t dataZid = zid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (xid >= dimensions.x || yid >= dimensions.y || dataZid >= dimensions.z)
	{
		return;
	}

	// Calculate the pointers for this block
	char* prevPsi = prevStep.ptr + prevStep.slicePitch * dataZid + prevStep.pitch * yid + sizeof(BlockPsis) * xid;
	BlockPsis* nextPsi = (BlockPsis*)(nextStep.ptr + nextStep.slicePitch * dataZid + nextStep.pitch * yid) + xid;

	// Update psi
	size_t dualNodeId = zid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)
	Complex3Vec prev = ((BlockPsis*)prevPsi)->values[dualNodeId];

	uint primaryFace = dualNodeId * FACE_COUNT;

	Complex3Vec H;
	H.s1 = make_double2(0, 0);
	H.s0 = make_double2(0, 0);
	H.s_1 = make_double2(0, 0);

#pragma unroll
	for (int i = 0; i < FACE_COUNT; ++i)
	{
		Complex3Vec otherBoundaryZeroCell = ((BlockPsis*)(prevPsi + lapInd[primaryFace].x))->values[lapInd[primaryFace].y];
		H.s1 += hodges[primaryFace] * (otherBoundaryZeroCell.s1 - prev.s1);
		H.s0 += hodges[primaryFace] * (otherBoundaryZeroCell.s0 - prev.s0);
		H.s_1 += hodges[primaryFace] * (otherBoundaryZeroCell.s_1 - prev.s_1);

		primaryFace++;
	}

	double normSq_s1 = prev.s1.x * prev.s1.x + prev.s1.y * prev.s1.y;
	double normSq_s0 = prev.s0.x * prev.s0.x + prev.s0.y * prev.s0.y;
	double normSq_s_1 = prev.s_1.x * prev.s_1.x + prev.s_1.y * prev.s_1.y;
	double normSq = normSq_s1 + normSq_s0 + normSq_s_1;
	
	// Add the total potential to Hamiltonian
	double3 localPos = getLocalPos(dualNodeId);
	double3 globalPos = make_double3(p0.x + block_scale * (xid * BLOCK_WIDTH_X + localPos.x),
		p0.y + block_scale * (yid * BLOCK_WIDTH_Y + localPos.y),
		p0.z + block_scale * (dataZid * BLOCK_WIDTH_Z + localPos.z));
	double totalPot = trap(globalPos) + (c0 + c2) * normSq;

	H.s1 += totalPot * prev.s1 + c2 * (-2.0 * normSq_s_1 * prev.s1 + star(prev.s_1) * prev.s0 * prev.s0 + 0 * prev.s_1);
	H.s0 += totalPot * prev.s0 + c2 * (star(prev.s0) * prev.s_1 * prev.s1 - normSq_s0 * prev.s0 + star(prev.s0) * prev.s1 * prev.s_1);
	H.s_1 += totalPot * prev.s_1 + c2 * (0 * prev.s1 + star(prev.s1) * prev.s0 * prev.s0 - 2.0 * normSq_s1 * prev.s_1);

	// Add the Zeeman term
	double3 B = magneticField(globalPos, 0);
	double2 Bxy = INV_SQRT_2 * make_double2(B.x, B.y);
	double2 Bxy_star = star(Bxy);
	
	H.s1 += (B.z * prev.s1 + Bxy_star * prev.s0);
	H.s0 += (Bxy * prev.s1 + Bxy_star * prev.s_1);
	H.s_1 += (Bxy * prev.s0 - B.z * prev.s_1);

	nextPsi->values[dualNodeId].s1 = prev.s1 - dt * make_double2(H.s1.x, H.s1.y);
	nextPsi->values[dualNodeId].s0 = prev.s0 - dt * make_double2(H.s0.x, H.s0.y);
	nextPsi->values[dualNodeId].s_1 = prev.s_1 - dt * make_double2(H.s_1.x, H.s_1.y);
};
#else
__global__ void update(PitchedPtr nextStep, PitchedPtr prevStep, int2* __restrict__ lapInd, double* __restrict__ hodges, uint3 dimensions, double block_scale, double3 p0, double dt)
{
	size_t xid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t yid = blockIdx.y * blockDim.y + threadIdx.y;
	size_t zid = blockIdx.z * blockDim.z + threadIdx.z;

	size_t dataZid = zid / VALUES_IN_BLOCK; // One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)

	// Exit leftover threads
	if (xid >= dimensions.x || yid >= dimensions.y || dataZid >= dimensions.z)
	{
		return;
	}

	// Calculate the pointers for this block
	char* prevPsi = prevStep.ptr + prevStep.slicePitch * dataZid + prevStep.pitch * yid + sizeof(BlockPsis) * xid;
	BlockPsis* nextPsi = (BlockPsis*)(nextStep.ptr + nextStep.slicePitch * dataZid + nextStep.pitch * yid) + xid;

	// Update psi
	size_t dualNodeId = zid % VALUES_IN_BLOCK; // Dual node id. One thread per every dual node so VALUES_IN_BLOCK threads per mesh block (on z-axis)
	Complex3Vec prev = ((BlockPsis*)prevPsi)->values[dualNodeId];

	uint primaryFace = dualNodeId * FACE_COUNT;

	Complex3Vec H;
	H.s1 = make_double2(0, 0);
	H.s0 = make_double2(0, 0);
	H.s_1 = make_double2(0, 0);

#pragma unroll
	for (int i = 0; i < FACE_COUNT; ++i)
	{
		H.s1 += hodges[primaryFace] * (((BlockPsis*)(prevPsi + lapInd[primaryFace].x))->values[lapInd[primaryFace++].y].s1 - prev.s1);
		H.s0 += hodges[primaryFace] * (((BlockPsis*)(prevPsi + lapInd[primaryFace].x))->values[lapInd[primaryFace++].y].s0 - prev.s0);
		H.s_1 += hodges[primaryFace] * (((BlockPsis*)(prevPsi + lapInd[primaryFace].x))->values[lapInd[primaryFace++].y].s_1 - prev.s_1);
	}

	double normSq_s1 = prev.s1.x * prev.s1.x + prev.s1.y * prev.s1.y;
	double normSq_s0 = prev.s0.x * prev.s0.x + prev.s0.y * prev.s0.y;
	double normSq_s_1 = prev.s_1.x * prev.s_1.x + prev.s_1.y * prev.s_1.y;
	double normSq = normSq_s1 + normSq_s0 + normSq_s_1;

	// Add the total potential to Hamiltonian
	double3 localPos = getLocalPos(dualNodeId);
	double3 globalPos = make_double3(p0.x + block_scale * (xid * BLOCK_WIDTH_X + localPos.x),
		p0.y + block_scale * (yid * BLOCK_WIDTH_Y + localPos.y),
		p0.z + block_scale * (dataZid * BLOCK_WIDTH_Z + localPos.z));
	double totalPot = trap(globalPos) + (c0 + c2) * normSq;

	H.s1 += totalPot * prev.s1 + c2 * (-2 * normSq_s_1 * prev.s1 + star(prev.s_1) * prev.s0 * prev.s0 + 0 * prev.s_1);
	H.s0 += totalPot * prev.s0 + c2 * (star(prev.s0) * prev.s_1 * prev.s1 - normSq_s0 * prev.s0 + star(prev.s0) * prev.s1 * prev.s_1);
	H.s_1 += totalPot * prev.s_1 + c2 * (0 * prev.s1 + star(prev.s1) * prev.s0 * prev.s0 - 2 * normSq_s1 * prev.s_1);

	// Add the Zeeman term
	double3 B = magneticField(globalPos, 0);
	double2 Bxy = INV_SQRT_2 * make_double2(B.x, B.y);
	double2 Bxy_star = star(Bxy);

	H.s1 += (B.z * prev.s1 + Bxy_star * prev.s0);
	H.s0 += (Bxy * prev.s1 + Bxy_star * prev.s_1);
	H.s_1 += (Bxy * prev.s0 - B.z * prev.s_1);

	nextPsi->values[dualNodeId].s1 += dt * make_double2(H.s1.y, -H.s1.x);
	nextPsi->values[dualNodeId].s0 += dt * make_double2(H.s0.y, -H.s0.x);
	nextPsi->values[dualNodeId].s_1 += dt * make_double2(H.s_1.y, -H.s_1.x);
};
#endif

//void energy_h(dim3 dimGrid, dim3 dimBlock, ddouble* energyPtr, PitchedPtr psi, PitchedPtr potentials, int2* lapInd, double* hodges, double g, uint3 dimensions, ddouble volume, size_t bodies)
//{
//	energy << <dimGrid, dimBlock >> > (energyPtr, psi, potentials, lapInd, hodges, g, dimensions, volume);
//	int prevStride = bodies;
//	while (prevStride > 1)
//	{
//		int newStride = prevStride / 2;
//		integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (energyPtr, newStride, ((newStride * 2) != prevStride));
//		prevStride = newStride;
//	}
//}

void normalize_h(dim3 dimGrid, dim3 dimBlock, ddouble* densityPtr, PitchedPtr psi, uint3 dimensions, size_t bodies, ddouble volume)
{
	density << <dimGrid, dimBlock >> > (densityPtr, psi, dimensions, volume);
	int prevStride = bodies;
	while (prevStride > 1)
	{
		int newStride = prevStride / 2;
		integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (densityPtr, newStride, ((newStride * 2) != prevStride));
		prevStride = newStride;
	}

	normalize << < dimGrid, dimBlock >> > (densityPtr, psi, dimensions);
}

void printDensity(dim3 dimGrid, dim3 dimBlock, ddouble* densityPtr, PitchedPtr psi, uint3 dimensions, size_t bodies, ddouble volume)
{
	density << <dimGrid, dimBlock >> > (densityPtr, psi, dimensions, volume);
	int prevStride = bodies;
	while (prevStride > 1)
	{
		int newStride = prevStride / 2;
		integrate << <dim3(std::ceil(newStride / 32.0), 1, 1), dim3(32, 1, 1) >> > (densityPtr, newStride, ((newStride * 2) != prevStride));
		prevStride = newStride;
	}
	ddouble hDensity = 0;
	checkCudaErrors(hipMemcpy(&hDensity, densityPtr, sizeof(ddouble), hipMemcpyDeviceToHost));

	std::cout << "Total density: " << hDensity << std::endl;
}

uint integrateInTime(const ddouble block_scale, const Vector3& minp, const Vector3& maxp, const ddouble iteration_period, const uint number_of_iterations)
{
	uint i, j, k, l;

	// find dimensions
	const Vector3 domain = maxp - minp;
	const uint xsize = uint(domain.x / (block_scale * BLOCK_WIDTH.x)) + 1;
	const uint ysize = uint(domain.y / (block_scale * BLOCK_WIDTH.y)) + 1;
	const uint zsize = uint(domain.z / (block_scale * BLOCK_WIDTH.z)) + 1;
	const Vector3 p0 = 0.5 * (minp + maxp - block_scale * Vector3(BLOCK_WIDTH.x * xsize, BLOCK_WIDTH.y * ysize, BLOCK_WIDTH.z * zsize));
	const double3 d_p0 = make_double3(p0.x, p0.y, p0.z);

	std::cout << xsize << ", " << ysize << ", " << zsize << std::endl;

	// compute discrete dimensions
	const uint bsize = VALUES_IN_BLOCK; // bpos.size(); // number of values inside a block
	const uint bxsize = (xsize + 1) * bsize; // number of values on x-row
	const uint bxysize = (ysize + 1) * bxsize; // number of values on xy-plane
	const uint ii0 = (IS_3D ? bxysize : 0) + bxsize + bsize; // reserved zeros in the beginning of value table
	const uint vsize = ii0 + (IS_3D ? zsize + 1 : zsize) * bxysize; // total number of values

	std::cout << "bsize: " << bsize << ", xsize: " << xsize << ", yszie: " << ysize << ", zsize: " << zsize << std::endl;
	uint64_t bodies = xsize * ysize * zsize * bsize;
	std::cout << "bodies = " << bodies << std::endl;

	// Initialize device memory
	size_t dxsize = xsize + 2; // One element buffer to both ends
	size_t dysize = ysize + 2; // One element buffer to both ends
	size_t dzsize = zsize + 2; // One element buffer to both ends
	hipExtent psiExtent = make_hipExtent(dxsize * sizeof(BlockPsis), dysize, dzsize);

	hipPitchedPtr d_cudaEvenPsi;
	hipPitchedPtr d_cudaOddPsi;
	checkCudaErrors(hipMalloc3D(&d_cudaEvenPsi, psiExtent));
	checkCudaErrors(hipMalloc3D(&d_cudaOddPsi, psiExtent));

	ddouble* d_energy;
	ddouble* d_density;
	checkCudaErrors(hipMalloc(&d_energy, bodies * sizeof(ddouble)));
	checkCudaErrors(hipMalloc(&d_density, bodies * sizeof(ddouble)));

	size_t offset = d_cudaEvenPsi.pitch * dysize + d_cudaEvenPsi.pitch + sizeof(BlockPsis);
	PitchedPtr d_evenPsi = { (char*)d_cudaEvenPsi.ptr + offset, d_cudaEvenPsi.pitch, d_cudaEvenPsi.pitch * dysize };
	PitchedPtr d_oddPsi = { (char*)d_cudaOddPsi.ptr + offset, d_cudaOddPsi.pitch, d_cudaOddPsi.pitch * dysize };

	// find terms for laplacian
	Buffer<int2> lapind;
	Buffer<ddouble> hodges;
	ddouble lapfac = -0.5 * getLaplacian(lapind, hodges, sizeof(BlockPsis), d_evenPsi.pitch, d_evenPsi.slicePitch) / (block_scale * block_scale);
	const uint lapsize = lapind.size() / bsize;
	ddouble lapfac0 = lapsize * (-lapfac);

	//std::cout << "lapsize = " << lapsize << ", lapfac = " << lapfac << ", lapfac0 = " << lapfac0 << std::endl;

	// compute time step size
	const uint steps_per_iteration = 1.0 / 0.000199999994947575; // uint(iteration_period * (maxpot + lapfac0)) + 1; // number of time steps per iteration period
	const ddouble dt = 0.000199999994947575; // iteration_period / ddouble(steps_per_iteration); // time step in time units

	std::cout << "steps_per_iteration = " << steps_per_iteration << std::endl;

	std::cout << "ALU operations per unit time = " << xsize * ysize * zsize * bsize * steps_per_iteration * FACE_COUNT << std::endl;

	for (int i = 0; i < hodges.size(); ++i) hodges[i] = -0.5 * hodges[i] / (block_scale * block_scale);

	int2* d_lapind;
	checkCudaErrors(hipMalloc(&d_lapind, lapind.size() * sizeof(int2)));

	ddouble* d_hodges;
	checkCudaErrors(hipMalloc(&d_hodges, hodges.size() * sizeof(ddouble)));

	// Initialize host memory
	size_t hostSize = dxsize * dysize * (zsize + 2);
	BlockPsis* h_evenPsi;// = new BlockPsis[dxsize * dysize * (zsize + 2)];
	BlockPsis* h_oddPsi;// = new BlockPsis[dxsize * dysize * (zsize + 2)];
	checkCudaErrors(hipHostMalloc(&h_evenPsi, hostSize * sizeof(BlockPsis)));
	checkCudaErrors(hipHostMalloc(&h_oddPsi, hostSize * sizeof(BlockPsis)));
	memset(h_evenPsi, 0, hostSize * sizeof(BlockPsis));
	memset(h_oddPsi, 0, hostSize * sizeof(BlockPsis));

	// initialize discrete field
	Random rnd(54363);
	for (k = 0; k < zsize; k++)
	{
		for (j = 0; j < ysize; j++)
		{
			for (i = 0; i < xsize; i++)
			{
				for (l = 0; l < bsize; l++)
				{
					const uint srcI = ii0 + k * bxysize + j * bxsize + i * bsize + l;
					const uint dstI = (k + 1) * dxsize * dysize + (j + 1) * dxsize + (i + 1);
					const Vector2 s1 = rnd.getUniformCircle();
					const Vector2 s0 = rnd.getUniformCircle();
					const Vector2 s_1 = rnd.getUniformCircle();
					h_evenPsi[dstI].values[l].s1 = make_double2(s1.x, s1.y);
					h_evenPsi[dstI].values[l].s0 = make_double2(s0.x, s0.y);
					h_evenPsi[dstI].values[l].s_1 = make_double2(s_1.x, s_1.y);
				}
			}
		}
	}

	hipPitchedPtr h_cudaEvenPsi = { 0 };
	hipPitchedPtr h_cudaOddPsi = { 0 };

	h_cudaEvenPsi.ptr = h_evenPsi;
	h_cudaEvenPsi.pitch = dxsize * sizeof(BlockPsis);
	h_cudaEvenPsi.xsize = d_cudaEvenPsi.xsize;
	h_cudaEvenPsi.ysize = d_cudaEvenPsi.ysize;

	h_cudaOddPsi.ptr = h_oddPsi;
	h_cudaOddPsi.pitch = dxsize * sizeof(BlockPsis);
	h_cudaOddPsi.xsize = d_cudaOddPsi.xsize;
	h_cudaOddPsi.ysize = d_cudaOddPsi.ysize;

	// Copy from host memory to device memory
	hipMemcpy3DParms evenPsiParams = { 0 };
	hipMemcpy3DParms oddPsiParams = { 0 };

	evenPsiParams.srcPtr = h_cudaEvenPsi;
	evenPsiParams.dstPtr = d_cudaEvenPsi;
	evenPsiParams.extent = psiExtent;
	evenPsiParams.kind = hipMemcpyHostToDevice;

	oddPsiParams.srcPtr = h_cudaOddPsi;
	oddPsiParams.dstPtr = d_cudaOddPsi;
	oddPsiParams.extent = psiExtent;
	oddPsiParams.kind = hipMemcpyHostToDevice;

	checkCudaErrors(hipMemcpy3D(&evenPsiParams));
	checkCudaErrors(hipMemcpy3D(&oddPsiParams));
	checkCudaErrors(hipMemcpy(d_lapind, &lapind[0], lapind.size() * sizeof(int2), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_hodges, &hodges[0], hodges.size() * sizeof(ddouble), hipMemcpyHostToDevice));

	// Clear host memory after data has been copied to devices
	hipDeviceSynchronize();
	lapind.clear();
	hodges.clear();
	hipHostFree(h_oddPsi);
#if !(SAVE_PICTURE || SAVE_VOLUME)
	hipHostFree(h_evenPsi);
#endif

	// Integrate in time
	uint3 dimensions = make_uint3(xsize, ysize, zsize);
	uint iter = 0;
	dim3 dimBlock(THREAD_BLOCK_X, THREAD_BLOCK_Y, THREAD_BLOCK_Z);
	dim3 dimGrid((xsize + THREAD_BLOCK_X - 1) / THREAD_BLOCK_X,
		(ysize + THREAD_BLOCK_Y - 1) / THREAD_BLOCK_Y,
		((zsize + THREAD_BLOCK_Z - 1) / THREAD_BLOCK_Z) * VALUES_IN_BLOCK);
#if SAVE_PICTURE || SAVE_VOLUME
	hipMemcpy3DParms evenPsiBackParams = { 0 };
	evenPsiBackParams.srcPtr = d_cudaEvenPsi;
	evenPsiBackParams.dstPtr = h_cudaEvenPsi;
	evenPsiBackParams.extent = psiExtent;
	evenPsiBackParams.kind = hipMemcpyDeviceToHost;
#endif

	const ddouble volume = (IS_3D ? block_scale : 1.0) * block_scale * block_scale * VOLUME;

#if COMPUTE_GROUND_STATE
	normalize_h(dimGrid, dimBlock, d_density, d_evenPsi, dimensions, bodies, volume);

	ddouble mu = 0;
	ddouble E = 1e20;
	auto Hpsi = d_oddPsi;
	while (true)
	{
		if ((iter % SAVE_FREQUENCY) == 0)
		{
			checkCudaErrors(hipMemcpy3D(&evenPsiBackParams));
			{
				// draw picture
				const ddouble INTENSITY = 20.0f;
				const int SIZE = 2;
				int width = dxsize * SIZE, height = dysize * SIZE;
				Picture pic(width, height);
				uint k = zsize / 2 + 1;
				for (uint j = 0; j < height; j++)
				{
					for (uint i = 0; i < width; i++)
					{
						const uint idx = k * dxsize * dysize + (j / SIZE) * dxsize + i / SIZE;
						double norm_s1 = sqrt(h_evenPsi[idx].values[0].s1.x * h_evenPsi[idx].values[0].s1.x + h_evenPsi[idx].values[0].s1.y * h_evenPsi[idx].values[0].s1.y);
						double norm_s0 = sqrt(h_evenPsi[idx].values[0].s0.x* h_evenPsi[idx].values[0].s0.x + h_evenPsi[idx].values[0].s0.y * h_evenPsi[idx].values[0].s0.y);
						double norm_s_1 = sqrt(h_evenPsi[idx].values[0].s_1.x * h_evenPsi[idx].values[0].s_1.x + h_evenPsi[idx].values[0].s_1.y * h_evenPsi[idx].values[0].s_1.y);

						pic.setColor(i, j, INTENSITY * Vector4(norm_s1, norm_s0, norm_s_1, 1.0));
					}
				}
				std::ostringstream picpath;
				picpath << "results/kuva" << iter << ".bmp";
				pic.save(picpath.str(), false);
			}
			{
				// draw picture
				const ddouble INTENSITY = 20.0f;
				const int SIZE = 2;
				int width = dxsize * SIZE, height = dysize * SIZE, depth = dzsize * SIZE;
				Picture pic(height, depth);
				uint j = height / 2;
				for (uint k = 0; k < depth; ++k)
				{
					for (uint i = 0; i < width; i++)
					{
						const uint idx = (k / SIZE) * dxsize * dysize + (j / SIZE) * dxsize + i / SIZE;
						double norm_s1 = sqrt(h_evenPsi[idx].values[0].s1.x * h_evenPsi[idx].values[0].s1.x + h_evenPsi[idx].values[0].s1.y * h_evenPsi[idx].values[0].s1.y);
						double norm_s0 = sqrt(h_evenPsi[idx].values[0].s0.x * h_evenPsi[idx].values[0].s0.x + h_evenPsi[idx].values[0].s0.y * h_evenPsi[idx].values[0].s0.y);
						double norm_s_1 = sqrt(h_evenPsi[idx].values[0].s_1.x * h_evenPsi[idx].values[0].s_1.x + h_evenPsi[idx].values[0].s_1.y * h_evenPsi[idx].values[0].s_1.y);

						pic.setColor(i, k, INTENSITY * Vector4(norm_s1, norm_s0, norm_s_1, 1.0));
					}
				}
				std::ostringstream picpath;
				picpath << "results/kuvaZX" << iter << ".bmp";
				pic.save(picpath.str(), false);
			}
			std::cout << iter << ": ";
			printDensity(dimGrid, dimBlock, d_density, d_evenPsi, dimensions, bodies, volume);
		}

		// Take an imaginary time step
		itp << <dimGrid, dimBlock >> > (d_oddPsi, d_evenPsi, d_lapind, d_hodges, dimensions, block_scale, d_p0, dt);
		// Normalize
		normalize_h(dimGrid, dimBlock, d_density, d_oddPsi, dimensions, bodies, volume);

		// Take an imaginary time step
		itp << <dimGrid, dimBlock >> > (d_evenPsi, d_oddPsi, d_lapind, d_hodges, dimensions, block_scale, d_p0, dt);
		// Normalize
		normalize_h(dimGrid, dimBlock, d_density, d_evenPsi, dimensions, bodies, volume);

		//energy_h(dimGrid, dimBlock, d_energy, d_evenPsi, d_pot, d_lapind, d_hodges, g, dimensions, volume, bodies);
		ddouble hDensity = 0;
		//ddouble hEnergy = 0;
		checkCudaErrors(hipMemcpy(&hDensity, d_density, sizeof(ddouble), hipMemcpyDeviceToHost));
		//checkCudaErrors(hipMemcpy(&hEnergy, d_energy, sizeof(ddouble), hipMemcpyDeviceToHost));

		//ddouble newMu = hEnergy / hDensity;
		//ddouble newE = hEnergy;
		//
		//std::cout << "Total density: " << hDensity << ", Total energy: " << hEnergy << ", mu: " << newMu << std::endl;

		

		//if (newE > E) break;
		//if (std::abs(mu - newMu) < 1e-4) break;
		if (iter > 100000) break;

		//mu = newMu;
		//E = newE;

		iter++;
	}

#else
	Text errorText;
	const uint time0 = clock();
	while (true)
	{
#if SAVE_PICTURE
		// draw picture
		const ddouble INTENSITY = 20.0f;
		const int SIZE = 2;
		int width = dxsize * SIZE, height = dysize * SIZE;
		Picture pic(width, height);
		k = zsize / 2 + 1;
		for (j = 0; j < height; j++)
		{
			for (i = 0; i < width; i++)
			{
				const uint idx = k * dxsize * dysize + (j / SIZE) * dxsize + i / SIZE;
				double norm = sqrt(h_evenPsi[idx].values[0].x * h_evenPsi[idx].values[0].x + h_evenPsi[idx].values[0].y * h_evenPsi[idx].values[0].y);
		
				pic.setColor(i, j, INTENSITY * Vector4(h_evenPsi[idx].values[0].x, norm, h_evenPsi[idx].values[0].y, 1.0));
			}
		}
		std::ostringstream picpath;
		picpath << "results/kuva" << iter << ".bmp";
		pic.save(picpath.str(), false);

		// print squared norm and error
		const Complex currentPhase = state.getPhase(iter * steps_per_iteration * dt);
		ddouble errorNormSq = 0;
		ddouble normsq = 0.0;
		Complex error(0.0, 0.0);
		for (k = 0; k < zsize; k++)
		{
			for (j = 0; j < ysize; j++)
			{
				for (i = 0; i < xsize; i++)
				{
					for (l = 0; l < bsize; l++)
					{
						const uint srcI = ii0 + k * bxysize + j * bxsize + i * bsize + l;
						const uint dstI = (k + 1) * dxsize * dysize + (j + 1) * dxsize + (i + 1);

						Complex evenPsi(h_evenPsi[dstI].values[l].x, h_evenPsi[dstI].values[l].y);
						normsq += evenPsi.normsq() * volume;
						error += (Psi0[srcI].con() * evenPsi) * volume;

						Complex groundTruth = currentPhase * Psi0[srcI];
						errorNormSq += (groundTruth - evenPsi).normsq();
					}
				}
			}
		}
		ddouble RMSE = sqrt(errorNormSq / (double)(zsize * ysize * xsize * bsize));
		ddouble errorAbs = abs(normsq - error.norm());
		std::cout << "normsq=" << normsq << " error=" << errorAbs << std::endl;
		errorText << RMSE << " ";
#endif

#if SAVE_VOLUME
		// save volume map
		const ddouble fmax = state.searchFunctionMax();
		const ddouble unit = 60000.0 / (bsize * fmax * fmax);
		Buffer<ushort> vol(dxsize * dysize * dzsize);
		for (k = 0; k < dzsize; k++)
		{
			for (j = 0; j < dysize; j++)
			{
				for (i = 0; i < dxsize; i++)
				{
					const uint idx = k * dxsize * dysize + j * dxsize + i;
					ddouble sum = 0.0;
					for (l = 0; l < bsize; l++)
					{
						sum += h_evenPsi[idx].values[0].x * h_evenPsi[idx].values[0].x + h_evenPsi[idx].values[0].y * h_evenPsi[idx].values[0].y;
					}
					sum *= unit;
					vol[idx] = (sum > 65535.0 ? 65535 : ushort(sum));
				}
			}
		}
		Text volpath;
		volpath << "volume" << iter << ".mhd";
		saveVolumeMap(volpath.str(), vol, dxsize, dysize, dzsize, block_scale * BLOCK_WIDTH);
#endif

		// finish iteration
		if (++iter > number_of_iterations) break;
		//++iter;
		//if (errorAbs > 0.01) break;

		// integrate one iteration
		std::cout << "Iteration " << iter << std::endl;
		for (uint step = 0; step < steps_per_iteration; step++)
		{
			// update odd values
			update << <dimGrid, dimBlock >> > (d_oddPsi, d_evenPsi, d_lapind, d_hodges, dimensions, block_scale, d_p0, dt);
			// update even values
			update << <dimGrid, dimBlock >> > (d_evenPsi, d_oddPsi, d_lapind, d_hodges, dimensions, block_scale, d_p0, dt);
		}

#if SAVE_PICTURE || SAVE_VOLUME
		// Copy back from device memory to host memory
		checkCudaErrors(hipMemcpy3D(&evenPsiBackParams));
#endif
	}
	errorText.save("results/errors.txt");

	std::cout << "iteration time = " << (1e-3 * (clock() - time0)) / number_of_iterations << std::endl;
	std::cout << "total time = " << 1e-3 * (clock() - time0) << std::endl;
#endif

	hipError_t err = hipGetLastError();
	if (err != hipSuccess)
	{
		fprintf(stderr, "Failed to launch kernels (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

	return 0;
}

int main(int argc, char** argv)
{
	const int number_of_iterations = 100;
	const ddouble iteration_period = 1.0;
	const ddouble block_scale = Lx / Nx;

	std::cout << "block_scale = " << block_scale << std::endl;
	std::cout << "iteration_period = " << iteration_period << std::endl;
	std::cout << "dual edge length = " << DUAL_EDGE_LENGTH * block_scale << std::endl;

	// integrate in time using DEC
	auto domainMin = Vector3(-Lx * 0.5, -Ly * 0.5, -Lz * 0.5);
	auto domainMax = Vector3(Lx * 0.5, Ly * 0.5, Lz * 0.5);
	integrateInTime(block_scale, domainMin, domainMax, iteration_period, number_of_iterations);

	return 0;
}
